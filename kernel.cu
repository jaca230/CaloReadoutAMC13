#include "hip/hip_runtime.h"
/**
 * @file    kernel.cu
 * @author  Tim Gorringe, Wes Gohn, Vladimir Tishchenko
 * @date    Last-Updated: Tue Oct 16 11:08:05 2018 (-0400)
 *          By : Wes Gohn
 *          Update #: 1063
 * @version $Id$
 * @copyright (c) new (g-2) collaboration
 *
 * 
 * 
 * @section Changelog
 * @verbatim
 * $Log$
 * @endverbatim
 */

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <linux/types.h>
//#include "cuPrintf.cu"

//#define DEBUG

#ifdef DEBUG
#define dbprintf(...) printf(__VA_ARGS__)
#else
#define dbprintf(...) 
#endif

// includes, project
#include <hip/hip_runtime.h>
#include "cuda_tools_g2.h"
#include "gpu_thread.h"
#include "midas.h"
#include "amc13_odb.h"

// ATF for template fitting
#include <cassert>
#include <vector>
#include <fstream>
#include <string>
#include "gpu_fit.hh"
#include "hip/hip_math_constants.h"

// N_SAMPLES_MAX (the maximum samples in individual waveform) is used in definition of structures in GPU_HIS_DATA  
// and GPU_AUX_DATA that are mapped to regions of gpu_odata that containing the histogram data and auxiliary data
#define N_SAMPLES_MAX 800000

#define USE_RIDER_FORMAT  1 // = 1 use Rider module/channel header trailers, = 0 dont use
#define N_RIDERCHANS    5 // unused?

// ADC type
#define ADC_TYPE       int16_t
#define ADC_MAX        2048

//cuda error checking
#define CUDA_ERROR_CHECK
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

// structure for histogram data
typedef struct s_gpu_his_data {
   int32_t  wf_hist[TQMETHOD_MAX*N_SAMPLES_MAX*N_DETECTORS_MAX];  // fill-summed waveform, 32-bit signed int array of size TQMETHOD_MAX*N_SAMPLES_MAX*N_DETECTORS_MAX
}  GPU_HIS_DATA;

// structure for auxiliary data
typedef struct s_gpu_aux_data {
  double   wf_sum[N_SAMPLES_MAX];        // sum waveform , double array of size N_SAMPLES_MAX
  double   pedestal[N_DETECTORS_MAX];        // calculated pedestal average,  double array of size N_DETECTORS_MAX
  int      island_pattern[N_SAMPLES_MAX*N_DETECTORS_MAX];// auxiliary array for island build
  int      islands_size;                 // total size of the array islands[]
  struct {
    int time;
    int detector;
    int length;
    int offset;
  } island_info[N_SAMPLES_MAX]; // structure array of size N_SAMPLES_MAX
} GPU_AUX_DATA;

// structure for output data
typedef struct s_gpu_out_data {
  int island_offset;                // used to record islands
  int n_islands;                    // number of islands found
  int CTAG ;                        //number of islands>2 GeV && t>50us
  int16_t islands[1];               // array of islands
} GPU_OUT_DATA;

/*
// energy calibration coefficients - for future?
__device__ static double A_calib[N_DETECTORS_MAX] = {
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0
  };
*/

// host, device parameters for GPU processing
__constant__ int DEV_N_samples[TQMETHOD_MAX]; 
int HOST_N_samples[TQMETHOD_MAX];
__constant__ int DEV_first_chop_sample[TQMETHOD_MAX]; 
int HOST_first_chop_sample[TQMETHOD_MAX];
__constant__ int DEV_last_chop_sample[TQMETHOD_MAX]; 
int HOST_last_chop_sample[TQMETHOD_MAX];
__constant__ int DEV_N_segments_x[TQMETHOD_MAX]; 
int HOST_N_segments_x[TQMETHOD_MAX];
__constant__ int DEV_N_segments_y[TQMETHOD_MAX]; 
int HOST_N_segments_y[TQMETHOD_MAX];
__constant__ int DEV_N_detectors[TQMETHOD_MAX]; 
int HOST_N_detectors[TQMETHOD_MAX];
__constant__ int DEV_rider_index[N_DETECTORS_MAX][TQMETHOD_MAX]; // detector element to electronics channel map
int HOST_rider_index[N_DETECTORS_MAX][TQMETHOD_MAX]; // detector element to electronics channel map
__constant__ int DEV_first_sample_index[N_DETECTORS_MAX][TQMETHOD_MAX]; // array offsets of first sample of each detector
int HOST_first_sample_index[N_DETECTORS_MAX][TQMETHOD_MAX]; // array offsets of first sample of each detector
__constant__ int DEV_N_presamples[TQMETHOD_MAX];
int HOST_N_presamples[TQMETHOD_MAX];
__constant__ int DEV_N_postsamples[TQMETHOD_MAX];
int HOST_N_postsamples[TQMETHOD_MAX];
__constant__ bool DEV_threshold_sign[TQMETHOD_MAX];
bool HOST_threshold_sign[TQMETHOD_MAX];
__constant__ int DEV_threshold[TQMETHOD_MAX];
int HOST_threshold[TQMETHOD_MAX];
__constant__ int DEV_pedestal_option[TQMETHOD_MAX];
int HOST_pedestal_option[TQMETHOD_MAX];
__constant__ int DEV_island_option[TQMETHOD_MAX];
int HOST_island_option[TQMETHOD_MAX];
__constant__ int DEV_global_pedestal[TQMETHOD_MAX];
int HOST_global_pedestal[TQMETHOD_MAX];
__constant__ bool DEV_hpedsubtract[TQMETHOD_MAX];
bool HOST_hpedsubtract[TQMETHOD_MAX];
__constant__ int DEV_hdecimation[TQMETHOD_MAX];
int HOST_hdecimation[TQMETHOD_MAX];
__constant__ int DEV_decimation[TQMETHOD_MAX];
int HOST_hfirstsample[TQMETHOD_MAX];
__constant__ int DEV_hfirstsample[TQMETHOD_MAX];
int HOST_hlastsample[TQMETHOD_MAX];
__constant__ int DEV_hlastsample[TQMETHOD_MAX];
int HOST_hrebinintervals[TQMETHOD_MAX];
__constant__ int DEV_hrebinintervals[TQMETHOD_MAX];
int HOST_hrebinincrement[TQMETHOD_MAX];
__constant__ int DEV_hrebinincrement[TQMETHOD_MAX];
int HOST_decimation[TQMETHOD_MAX];
__constant__ int DEV_hoffset[TQMETHOD_MAX];
int HOST_hoffset[TQMETHOD_MAX];
__constant__ bool DEV_useindividualthresholds[TQMETHOD_MAX];
bool HOST_useindividualthresholds[TQMETHOD_MAX];
__constant__ int DEV_thresholdvalues[N_DETECTORS_MAX][TQMETHOD_MAX];
int HOST_thresholdvalues[N_DETECTORS_MAX][TQMETHOD_MAX];
__constant__ bool DEV_thresholdpolarities[N_DETECTORS_MAX][TQMETHOD_MAX];
bool HOST_thresholdpolarities[N_DETECTORS_MAX][TQMETHOD_MAX];
__constant__ int DEV_nfitislands[TQMETHOD_MAX];
int HOST_nfitislands[TQMETHOD_MAX];
__constant__ int DEV_fit_threshold[TQMETHOD_MAX];
int HOST_fit_threshold[TQMETHOD_MAX];
__constant__ int DEV_minfittime[TQMETHOD_MAX];
int HOST_minfittime[TQMETHOD_MAX];
__constant__ int DEV_ctag_threshold[TQMETHOD_MAX];
int HOST_ctag_threshold[TQMETHOD_MAX];
__constant__ int DEV_ctag_time_cut[TQMETHOD_MAX];
int HOST_ctag_time_cut[TQMETHOD_MAX];

bool HOST_TMask_window[TQMETHOD_MAX];
__constant__ bool DEV_TMask_window[TQMETHOD_MAX];
int HOST_mask_min[TQMETHOD_MAX];
__constant__ int DEV_mask_min[TQMETHOD_MAX];
int HOST_mask_max[TQMETHOD_MAX];
__constant__ int DEV_mask_max[TQMETHOD_MAX];
int HOST_mask_prescale[TQMETHOD_MAX];
__constant__ int DEV_mask_prescale[TQMETHOD_MAX];

bool HOST_save_full_calo[TQMETHOD_MAX];
__constant__ bool DEV_save_full_calo[TQMETHOD_MAX];
int HOST_fill_type[TQMETHOD_MAX];
__constant__ int DEV_fill_type[TQMETHOD_MAX];

int HOST_hsize[TQMETHOD_MAX];
__constant__ int DEV_hsize[TQMETHOD_MAX];
int HOST_fill_seq[TQMETHOD_MAX];
__constant__ int DEV_fill_seq[TQMETHOD_MAX];

bool HOST_save_xtal_border[TQMETHOD_MAX];
__constant__ bool DEV_save_xtal_border[TQMETHOD_MAX];

int HOST_fit_prescale_factor[TQMETHOD_MAX];
__constant__ int DEV_fit_prescale_factor[TQMETHOD_MAX];

/*
seems nuts but last copy of variable from host to device fails so added a dummy variable / copy
*/
__constant__ int DEV_dummy[TQMETHOD_MAX];
int HOST_dummy[TQMETHOD_MAX];

int HOST_firstsampledummy[2], HOST_lastsampledummy[2];

__device__ phaseMap d_phase_maps[N_DETECTORS_MAX];
__device__ pulseTemplate d_templates[N_DETECTORS_MAX];

#if 0
/** 
 * Makes the distribution of ADC samples (fills ADC arrays
 * in GPU_AUX_DATA)
 * Histogramming is a bad task for GPU
 * Need a better solution
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
// kernel_wf_make_ADC is no longer used
void kernel_wf_make_ADC(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
 // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  int sum, idet;
  for (idet=0; idet<DEV_N_detectors[itq]; idet++)
    {
      int i = tid + bid*num_threads; 
      while ( i < DEV_N_samples[itq] )
	{
	  ADC_TYPE adc = gpu_idata[idet*DEV_N_samples[itq] + i];
	  atomicAdd( &(data->ADC[idet][adc]), 1);
	  sum += adc;
	  i += blockDim.x * gridDim.x;
	}
    }

}
#endif

#if 0
__global__
void kernel_print_map(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
 // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tidx = threadIdx.x;
  const unsigned int tidy = threadIdx.y;

  /*
  //cuPrintf doesn't work on the Fermilab system

  int idet = tidx+N_SEGMENTS_X*tidy;
  cuPrintf("kernel_print_map: thread.x %d, thread.y %d, module %d, channel %d\n", 
  	 tidx, tidy, SegXYtoRiderModu[tidx][tidy], SegXYtoRiderChan[tidx][tidy]);
  cuPrintf("kernel_print_map: structure nrmh %d, nrmt %d, nrch %d, nrct %d\n", 
  	   RiderParams.nrmhwords, RiderParams.nrmtwords, RiderParams.nrchwords, RiderParams.nrctwords);
  cuPrintf("kernel_print_map: DEV_first_sample_index %d dev_thres %d dev_decimation %d nrmh %d, nrmt %d, nrch %d, nrct %d\n", 
  	   DEV_first_sample_index[idet], DEV_threshold[itq], DEV_decimation[itq], NRMH_WORDS, NRMT_WORDS, NRCH_WORDS, NRCT_WORDS );
  cuPrintf("kernel_print_map: xsegment %d, ysegment %d, idet %d,  pedestal %f, first sample %i, next sample %i\n", 
  	   tidx, tidy, idet, auxdata->pedestal[idet], gpu_idata[DEV_first_sample_index[idet][itq]], gpu_idata[DEV_first_sample_index[idet][itq]+1] );
  */
}
#endif

/** 
 * Make a fill-by-fill sum of waveforms in each detector / segment
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_be64tole16(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  int sampletimesdetector_nr = 4 * ( tid + bid*num_threads ); 
  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ) { 
    
    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];

    // re-order the bytes within 2-byte words
    unsigned int lobyte, hibyte, four2bytewords[4];
    for (int iByteReorder = 0; iByteReorder  < 4; iByteReorder++ ){
      hibyte = (gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr + iByteReorder] & 0xff00) >> 8;
      lobyte = (gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr + iByteReorder] & 0xff);
      
      four2bytewords[iByteReorder] = lobyte << 8 | hibyte;
    }
    // re-order the 2-byte words within 8-byte words
    for (int iByteReorder = 0; iByteReorder  < 4; iByteReorder++ ){
      gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr + (3 - iByteReorder) ] = four2bytewords[iByteReorder];
    }
  }
}

/** 
 * Make a fill-by-fill sum of waveforms in each detector / segment
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_fillsum(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  // original index - has problem of serial rebinning and slow function
  int sampletimesdetector_nr = tid + bid*num_threads; 

  // maximum rebinning factor
  int rb = DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)(DEV_hrebinintervals[itq] -1) );
  // new index - avoid problem of serial rebinning and slow function
  //int sampletimesdetector_nr = (tid + bid*num_threads)/rb + rb*(tid%rb); 

  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ){

    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];  // for given detector
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];  // detector identifier
    
    // check sample index is between first / last samples in histogram
    if ( sample_nr >= (DEV_hfirstsample[itq] - 1) && sample_nr < DEV_hlastsample[itq] ) {
      
      // using calo map with time decimation and memory offset for multi TQ-methods
      ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];

      // subtract pedestal if desired
      if (DEV_hpedsubtract[itq]) adc = adc - auxdata->pedestal[idet];
     
      // account for first sample / last sample bookending
      int modsampletimesdetector_nr = (sample_nr - DEV_hfirstsample[itq] + 1) + idet * ( DEV_hlastsample[itq] - DEV_hfirstsample[itq] + 1);

      // calculate the number of samples in each rebinning interval
      // e.g. 512 /4 = 128
      int nsamplesperinterval = (  DEV_N_detectors[itq] * ( DEV_hlastsample[itq] - DEV_hfirstsample[itq] + 1) ) / DEV_hrebinintervals[itq];

      // calculate the index of the  sub-hist for the particular sample
      // e.g. 0 / 128 = 0
      int isubhistindex =  modsampletimesdetector_nr / nsamplesperinterval;
 
      // calculate the rebinning factor  of the corresponding sub-hist for the particular sample
      // eg. 2 * 2^0 
      // note pow() only defined in cuda for floating point
      int isubhistrebin = DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)isubhistindex );

      // calculate the bin number in the ith sub-hist with rebinning factor (DEV_hdecimation[itq] + i*DEV_rebinintervals[itq]) 
      int isubhistbin = ( modsampletimesdetector_nr % nsamplesperinterval ) / isubhistrebin;

      // calculate the sub-histogram offset within the full, variable bin-width histogram
      int io, isubhistoffset = 0;
      for (io = 0; io < isubhistindex; io++) isubhistoffset += nsamplesperinterval / ( DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)io ) );

      // combine sub-hist bin and sub-hist offset with memory offset for multiple TQ methods
      int index = DEV_hoffset[itq] + isubhistbin  + isubhistoffset;


      /*Block below temporary test of logarithmic histograms.
	If we decide to use it, add flag in ODB to toggle on/off
	Nothing was changed except what is inside if statement, so just delete to go back to how it was.
      */
      int log_hist=0;
      if(log_hist){
	int Nbins = (DEV_hlastsample[itq]-DEV_hfirstsample[itq])/DEV_hdecimation[itq];
	float b = log10f(DEV_hlastsample[itq]/DEV_hfirstsample[itq])/(DEV_hlastsample[itq]-DEV_hfirstsample[itq]);
	float a = DEV_hlastsample[itq]/exp10f(b*DEV_hlastsample[itq]);
	float samp_unscaled=sample_nr*(DEV_hlastsample[itq]-DEV_hfirstsample[itq])/Nbins + DEV_hfirstsample[itq];
	float logbin = a*exp10f(b*samp_unscaled);
	index = DEV_hoffset[itq] + idet*logbin;
      }

      // atomicAdd( &(hisdata->wf_hist[sampletimesdetector_nr / DEV_hdecimation[itq]]), adc );
      atomicAdd( &(hisdata->wf_hist[index]), adc );  

      // debug printout
      //printf("kernel_wf_fillsum: TQ %i, sample_nr %i, detector_nr %i,  sampletimesdetector_nr %i, sub-hist rebin %i, sub-hist bin %i, sub-hist index %i, sub-hist offset %i, initial rebin %i, rebin intervals %i, rebin multiplier %i, samples / interval %i\n", 
      //	     itq, sample_nr, idet, modsampletimesdetector_nr, isubhistrebin, isubhistbin, isubhistindex, isubhistoffset, DEV_hdecimation[itq], DEV_hrebinintervals[itq], DEV_hrebinincrement[itq], nsamplesperinterval );
      
    } //if sample index is between first / last samples in histogram
    
  } // protection against exceeding sample array

}

/** 
 * Make a fill-by-fill sum of waveforms in each detector / segment (launched for each bin of rebinned histo).
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_fillsum2(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq, int isubhist, int ifillnum)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  // This kernel function is launched for each bin of a particular sub-histogram.  

  // sub-histogram bin index (i.e. bin of sub-histogram)
  int isubbin = tid + bid*num_threads; 

  // calculate the number of raw samples in each rebinning interval, 
  // number of detectors * total raw sample range (first sample to last sample) / number of rebin intervals
  int nsamplesperinterval = ( DEV_N_detectors[itq] * ( DEV_hlastsample[itq] - DEV_hfirstsample[itq] + 1) ) / DEV_hrebinintervals[itq];

  // calculate the number of raw samples per individual detector in each rebinning interval
  // total raw sample range (first sample to last sample) / number of rebin intervals
  int nsamplesperintervalperdetector = ( DEV_hlastsample[itq] - DEV_hfirstsample[itq] + 1) / DEV_hrebinintervals[itq];

  // calculate the rebinning factor  of the corresponding sub-hist for the particular sample
  // initial rebin factor * (rebin mulitiplier)^(subhist index)
  int isubhistrebin = DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)isubhist );

  // calculate the number of bins for sub-histogram
  // total number of raw samples per sub-histogram / rebinning factor for sub-histogram
  int nsubbins = nsamplesperinterval / isubhistrebin;

  // calculate the number of bins per detector for sub-histogram
  // number of raw samples per individual detector per sub-histogram / rebinning factor for sub-histogram
  int nsubbinsperdetector = nsubbins / DEV_N_detectors[itq];

  // calculate sample offset for ith sub-histogram
  int isubhistoffset = nsubbinsperdetector * isubhist;

  // calculate detector index
  int idet = isubbin / nsubbinsperdetector;  

  if ( isubbin < nsubbins ){

    int adc = 0, firstrawbin = 0, sample_nr;

    // loop over raw sample bins corresponding to  particular sub-histogram bin
    firstrawbin = (( isubbin%nsubbinsperdetector ) + isubhistoffset ) * isubhistrebin;
    for (sample_nr = firstrawbin; sample_nr < firstrawbin + isubhistrebin; sample_nr++){

      // using calo map with time decimation and memory offset for multi TQ-methods
      adc += gpu_idata[ DEV_first_sample_index[idet][itq] + (DEV_hfirstsample[itq]-1) + sample_nr];

      // subtract pedestal if desired
      if (DEV_hpedsubtract[itq]) adc -= auxdata->pedestal[idet];
    }

    int io, ihistoffset = 0;
    int ioff1 = 0, ioff2 = 0;

    // calculate the sub-histogram offset within the full, variable bin-width histogram, with subhist1, subhist2, ... sequence
    //for (io = 0; io < isubhist; io++) ihistoffset += nsamplesperinterval / ( DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)io ) );

    // calculate the sub-histogram offset within the full, variable bin-width histogram, with det1, det2, ... sequence
    for (io = 0; io < DEV_hrebinintervals[itq]; io++) ioff1 += nsamplesperintervalperdetector / ( DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)io ) );
    for (io = 0; io < isubhist; io++) ioff2 += nsamplesperintervalperdetector / ( DEV_hdecimation[itq] * pow( (double)DEV_hrebinincrement[itq], (double)io ) );

    ihistoffset = idet * ioff1 + ioff2;    

    // combine sub-hist bin and sub-hist offset with memory offset for multiple TQ methods

    int dindex = ihistoffset + isubbin%nsubbinsperdetector;
    int index = DEV_hoffset[itq] + dindex;

    // calculate the fill sequence offset and modify index for filling hisogram
    int iseqnum = ifillnum % DEV_fill_seq[itq]; 
    dindex = ( DEV_hsize[itq] * iseqnum ) / DEV_fill_seq[itq];
    index += dindex;
    //if (isubbin == 1) printf("kernel_wf_fillsum2, adc %f, index %i, dindex %i, ifillnum %i, iseqnum %i, DEV_hsize[itq] %i,  DEV_fill_seq[itq] %i\n", adc, index, dindex, ifillnum, iseqnum, DEV_hsize[itq],  DEV_fill_seq[itq]); 

    // increment histogram array
    hisdata->wf_hist[index] += adc;

    //printf("tid %i, bid %i, num_threads %i, itq %i, isubhist %i, isubbin %i, nsubbins %i, subhistoffset %i, ioff1 %i, ioff2 %i, ihistoffset %i, rebin %i, sample %i, detector %i, dindex %i, adc sum %i\n", 
    //	   tid, bid, num_threads, itq, isubhist, isubbin, nsubbins, isubhistoffset, ioff1, ioff2, ihistoffset, isubhistrebin, firstrawbin, idet, dindex, adc );
  }

}


/** 
 * Make a sum of waveforms and identifiy triggers based on calo sum
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_sum(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq) 
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  int sample_nr = tid + bid*num_threads; 

  while ( sample_nr < DEV_N_samples[itq] )
    {
      double adc_sum = 0;
      unsigned int idet;
      // calc pedestal-subtracted calo sum
      for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	{

	  ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];
	  adc_sum += (adc - auxdata->pedestal[idet]); 
	}      
      auxdata->wf_sum[sample_nr] = adc_sum;

      // only create a trigger if sample is within the chopping time window of first_chop_sample to last_chop_sample  
      if (sample_nr < DEV_first_chop_sample[itq] || sample_nr > DEV_last_chop_sample[itq] ) return;

      // positive-going global threashold on calo sum
      if ( DEV_island_option[itq]==1 && DEV_threshold_sign[itq] && adc_sum > DEV_threshold[itq] ) 
	{
	  //printf("kernel_wf_sum trigger : %f , sample %d\n", adc_sum, sample_nr);
	  auxdata->island_pattern[sample_nr] = 1;
	}
      // negative-going global threashold on calo sum
      if ( DEV_island_option[itq]==1 && !DEV_threshold_sign[itq] && adc_sum < DEV_threshold[itq] )
	{
	  //printf("kernel_wf_sum trigger : %f , sample %d\n", adc_sum, sample_nr);
	  auxdata->island_pattern[sample_nr] = 1;
	}
      
      // periodic trigger for testing, debugging 
      //const int trigger_period = 50000;
      const int trigger_period = 30000;
      if(DEV_island_option[itq]==0 && sample_nr%trigger_period == 0 )
	{      
	  //printf("kernel_wf_sum trigger : %f , sample %d\n", adc_sum, sample_nr);
	  auxdata->island_pattern[sample_nr] = 1;
	}
      
      sample_nr += blockDim.x * gridDim.x;
    }

}

/**                                                                                                                                                               
 * find pulses in waveform                                                                                                                                 
 *                                                                        
 * @param gpu_idata                                                                                                                                
 * @param gpu_odata                                                                                                                       
 */
__global__
void kernel_wf_fittimes(ADC_TYPE *gpu_idata, pulseFinderResultCollection* resultcol, int itq)
{
  unsigned int tracelen = DEV_N_samples[itq];
  for (uint segment_num = 0; segment_num < DEV_N_detectors[itq]; ++segment_num) {
    // find index based on global id
    uint sample_num = blockIdx.x * blockDim.x + threadIdx.x;

    // don't continue if your trace_index is out of bounds for pulse fitting
    if (sample_num < DEV_minfittime[itq] ||
        sample_num >= tracelen - (SAMPLESPERFIT - PEAKINDEXINFIT - 1)) {
      return;
    }

    ADC_TYPE* trace = gpu_idata + DEV_first_sample_index[segment_num][itq];
//    ADC_TYPE polarity = DEV_thresholdpolarities[DEV_rider_index[segment_num][itq]][itq] ? 1 : -1;
    // change to use global polarity variable. Fang Han.
    ADC_TYPE polarity = DEV_threshold_sign[itq] ? 1 : -1;


    // we need the samples at this point and surrounding (left, middle, right)
    short sample_val = trace[sample_num];
    short m = polarity * sample_val;
    // if this sample is a local minimum and is over threshold, record it
    if (m > polarity * DEV_fit_threshold[itq]) {
      short l = polarity * trace[sample_num - 1];
      short r = polarity * trace[sample_num + 1];

      // must be local minimum, but since we have digital ADC values
      // we must allow for max to equal sample on left, but not on right
      // if we allow max sample to equal right, we will fit same pulse twice
      if ((m >= l) && (m > r)) {
        uint pulse_index = atomicAdd(&resultcol[segment_num].nPulses, 1);
        if (pulse_index < OUTPUTARRAYLEN) {
          // find pulse time, phase
          // first calculate pseudo time
          float numerator = l - m;
          float denominator = r - m;
          // denominator can't be zero because m > r
          float ptime = 2.0 / HIP_PI_F * atan(numerator / denominator);

          // next interpolate time map table
          float where = ptime * (NPOINTSPHASEMAP - 1);
          int low_index = floor(where);
          float weight_heigh = where - low_index;
          float real_time;
          // check for out of bounds
          if (low_index < 0) {
            real_time = 0;
          } else if (low_index >= NPOINTSPHASEMAP - 1) {
            real_time = 1.0;
          } else {
            // do the interpolation
            real_time =
                d_phase_maps[segment_num].table[low_index] *
                    (1 - weight_heigh) +
                d_phase_maps[segment_num].table[low_index + 1] * weight_heigh;
          }

          // record time, phase, peak index, peak value
          float time_offset = d_phase_maps[segment_num].timeOffset;
          resultcol[segment_num].fit_results[pulse_index].time =
              sample_num + real_time - 0.5 - time_offset;
          resultcol[segment_num].fit_results[pulse_index].phase = 1 - real_time;
          resultcol[segment_num].fit_results[pulse_index].peak_index =
              sample_num;
          resultcol[segment_num].fit_results[pulse_index].peak_value = sample_val;
        } else {
          // beyond limit for number of pulses we're trying to fit
          atomicSub(&resultcol[segment_num].nPulses, 1);
        }
      }
    }
  }
}

/**                                                                                                                                                               
 * fit pulses found in kernel_wf_fittimes                                                                                                                                
 *                                                                        
 * @param gpu_idata                                                                                                                                
 * @param gpu_odata                                                                                                                       
 */
// according to info on nvidia.com, no need to explicitly synchronize threads
// within groups of 32
// because warps are 32 threads and instructions in warp are always synchronous
const int PULSESPERBLOCK = 4;
const int FIT_THREADSPERBLOCK = PULSESPERBLOCK * SAMPLESPERFIT;
#if 0
__global__
void kernel_wf_fitenergies(ADC_TYPE *gpu_idata, pulseFinderResultCollection* resultcol, int itq) {
  // arrays for accumulation
  __shared__ float tSum[FIT_THREADSPERBLOCK];
  __shared__ float dSum[FIT_THREADSPERBLOCK];
  __shared__ float dDotT[FIT_THREADSPERBLOCK];
  __shared__ float tDotT[FIT_THREADSPERBLOCK];

  unsigned int segment_num = blockIdx.y;
  unsigned int pulse_num =
      blockIdx.x * PULSESPERBLOCK + threadIdx.x / SAMPLESPERFIT;

  // return asap if this pulse doesn't exit
  if ((pulse_num >= OUTPUTARRAYLEN) ||
      (pulse_num >= resultcol[segment_num].nPulses)) {
    return;
  }

  // step one: read needed inputs from resultcol
  float phase = resultcol[segment_num].fit_results[pulse_num].phase;
  unsigned int start_sample = resultcol[segment_num].fit_results[pulse_num].peak_index - PEAKINDEXINFIT; 

  ADC_TYPE* fit_trace = gpu_idata + DEV_first_sample_index[segment_num][itq] + start_sample;

  // step two: read in template values for this phase and sample num
  unsigned int sample_index = threadIdx.x % SAMPLESPERFIT;
  float phase_loc = phase * POINTSPERSAMPLE;
  int phase_index = floor(phase_loc);
  float weight_high = phase_loc - phase_index;
  // make sure we're in bounds
  if (phase_index < 0) {
    phase_index = 0;
    weight_high = 0;
  } else if (phase_index >= POINTSPERSAMPLE) {
    phase_index = POINTSPERSAMPLE - 1;
    weight_high = 1;
  }
  unsigned int low_index = phase_index * SAMPLESPERFIT + sample_index;
  unsigned int high_index = low_index + SAMPLESPERFIT;
  float low_value = d_templates[segment_num].table[low_index];
  float high_value = d_templates[segment_num].table[high_index];

  // step 2.5 evaluate template
  float t_i = low_value * (1 - weight_high) + high_value * weight_high;

  // step three : read in pulse value
  float d_i = fit_trace[sample_index];

  // step four: prepare accumulation/reduction arrays
  tSum[threadIdx.x] = t_i;
  dSum[threadIdx.x] = d_i;
  dDotT[threadIdx.x] = d_i * t_i;
  tDotT[threadIdx.x] = t_i * t_i;

  // step five: accumulate, note that explicit synchronization
  // is not required because all accumulation is done within a warp
  // it seems like this stops working if the if and for are inverted, so don't
  // do that
  for (unsigned int stride = 16; stride >= 1; stride /= 2) {
    if (sample_index < 16) {
      tSum[threadIdx.x] += tSum[threadIdx.x + stride];
      dSum[threadIdx.x] += dSum[threadIdx.x + stride];
      dDotT[threadIdx.x] += dDotT[threadIdx.x + stride];
      tDotT[threadIdx.x] += tDotT[threadIdx.x + stride];
    }
  }

  // step six : calculate pedestal, energy
  // read final accumulated results
  int result_index = (threadIdx.x / SAMPLESPERFIT) * SAMPLESPERFIT;
  float tSumFinal = tSum[result_index];
  float dSumFinal = dSum[result_index];
  float dDotTFinal = dDotT[result_index];
  float tDotTFinal = tDotT[result_index];

  float denomRecip = 1.0 / (tSumFinal * tSumFinal - SAMPLESPERFIT * tDotTFinal);

  float energy =
      denomRecip * (dSumFinal * tSumFinal - SAMPLESPERFIT * dDotTFinal);
  float pedestal =
      denomRecip * (dDotTFinal * tSumFinal - dSumFinal * tDotTFinal);

  // step seven: load partial chi^2s into shared memory
  __shared__ float chi2sum[FIT_THREADSPERBLOCK];
  float residual_i = d_i - energy * t_i - pedestal;
  chi2sum[threadIdx.x] = residual_i * residual_i;

  // step eight: accumulate partial chi2s
  for (unsigned int stride = 16; stride >= 1; stride /= 2) {
    if (sample_index < 16) {
      chi2sum[threadIdx.x] += chi2sum[threadIdx.x + stride];
    }
  }
 
  // final step: record results

  // force energy positive
  if (energy < 0) {
    energy = energy * -1;
  }
  //if (sample_index == 0) {
  if(sample_index == PEAKINDEXINFIT){
    resultcol[segment_num].fit_results[pulse_num].energy = d_i;//energy;
    resultcol[segment_num].fit_results[pulse_num].pedestal = t_i;//pedestal;
    resultcol[segment_num].fit_results[pulse_num].chi2 = chi2sum[threadIdx.x];
  }
}
#endif

// A. Fienberg 2020-Dec-09
// bug that caused incorrect energy and pedestal calculations found to be
// caused by thread synchronization, possibly from updated driver/compiler.
// Added explicit thread synchronization bewtween array reduction steps (__syncthreads()).
// Pedestal and energy are correct now.
__global__
void kernel_wf_fitenergies(ADC_TYPE *gpu_idata, pulseFinderResultCollection* resultcol, int itq) {
  // arrays for accumulation
  __shared__ float tSum[FIT_THREADSPERBLOCK];
  __shared__ float dSum[FIT_THREADSPERBLOCK];
  __shared__ float dDotT[FIT_THREADSPERBLOCK];
  __shared__ float tDotT[FIT_THREADSPERBLOCK];
  unsigned int segment_num = blockIdx.y;
  unsigned int pulse_num =
      blockIdx.x * PULSESPERBLOCK + threadIdx.x / SAMPLESPERFIT;
  // step one: read needed inputs from resultcol
  float phase;
  unsigned int start_sample;
  bool valid;
  if ((pulse_num >= OUTPUTARRAYLEN) ||
      (pulse_num >= resultcol[segment_num].nPulses)) {
    phase = 0;
    start_sample = 0;
    valid = false;
  } else {
    phase = resultcol[segment_num].fit_results[pulse_num].phase;
    start_sample = resultcol[segment_num].fit_results[pulse_num].peak_index - PEAKINDEXINFIT;
    valid = true;
  }
  ADC_TYPE* fit_trace = gpu_idata + DEV_first_sample_index[segment_num][itq] + start_sample;
  // step two: read in template values for this phase and sample num
  unsigned int sample_index = threadIdx.x % SAMPLESPERFIT;
  float phase_loc = phase * POINTSPERSAMPLE;
  int phase_index = floor(phase_loc);
  float weight_high = phase_loc - phase_index;
  // make sure we're in bounds
  if (phase_index < 0) {
    phase_index = 0;
    weight_high = 0;
  } else if (phase_index >= POINTSPERSAMPLE) {
    phase_index = POINTSPERSAMPLE - 1;
    weight_high = 1;
  }
  unsigned int low_index = phase_index * SAMPLESPERFIT + sample_index;
  unsigned int high_index = low_index + SAMPLESPERFIT;
  float low_value = d_templates[segment_num].table[low_index];
  float high_value = d_templates[segment_num].table[high_index];
  // step 2.5 evaluate template
  float t_i = low_value * (1 - weight_high) + high_value * weight_high;
  // step three : read in pulse value
  float d_i = fit_trace[sample_index];
  // step four: prepare accumulation/reduction arrays
  tSum[threadIdx.x] = t_i;
  dSum[threadIdx.x] = d_i;
  dDotT[threadIdx.x] = d_i * t_i;
  tDotT[threadIdx.x] = t_i * t_i;
  // step five: accumulate
  __syncthreads();
  for (unsigned int stride = SAMPLESPERFIT/2; stride >= 1; stride /= 2) {
    if (sample_index < stride) {
      tSum[threadIdx.x] += tSum[threadIdx.x + stride];
      dSum[threadIdx.x] += dSum[threadIdx.x + stride];
      dDotT[threadIdx.x] += dDotT[threadIdx.x + stride];
      tDotT[threadIdx.x] += tDotT[threadIdx.x + stride];
    }
    __syncthreads();
  }
  // step six : calculate pedestal, energy
  // read final accumulated results
  int result_index = (threadIdx.x / SAMPLESPERFIT) * SAMPLESPERFIT;
  float tSumFinal = tSum[result_index];
  float dSumFinal = dSum[result_index];
  float dDotTFinal = dDotT[result_index];
  float tDotTFinal = tDotT[result_index];
  float denomRecip = 1.0 / (tSumFinal * tSumFinal - SAMPLESPERFIT * tDotTFinal);
  float energy =
      denomRecip * (dSumFinal * tSumFinal - SAMPLESPERFIT * dDotTFinal);
  float pedestal =
      denomRecip * (dDotTFinal * tSumFinal - dSumFinal * tDotTFinal);
  // step seven: load partial chi^2s into shared memory
  __shared__ float chi2sum[FIT_THREADSPERBLOCK];
  float residual_i = d_i - energy * t_i - pedestal;
  chi2sum[threadIdx.x] = residual_i * residual_i;
  // step eight: accumulate partial chi2s
  __syncthreads();
  for (unsigned int stride = SAMPLESPERFIT/2; stride >= 1; stride /= 2) {
    if (sample_index < stride) {
      chi2sum[threadIdx.x] += chi2sum[threadIdx.x + stride];
    }
    __syncthreads();
  }
  // final step: record results
  // force energy positive
  if (energy < 0) {
    energy = energy * -1;
  }
  if (valid && sample_index == 0) {
    resultcol[segment_num].fit_results[pulse_num].energy = energy;
    resultcol[segment_num].fit_results[pulse_num].pedestal = pedestal;
    resultcol[segment_num].fit_results[pulse_num].chi2 = chi2sum[threadIdx.x];
  }
}


/** 
 * Find the triggers in waveforms
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_trigger(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq, bool tmask_prescale)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id, block id, .. to define the sample index
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  const unsigned int num_threads = blockDim.x;

  // used for trigger option = 4 tht records trigger island and Tcyc delayed island
  int TcycClockTicks = 120;
 
  int sample_nr = tid + bid*num_threads;

  // only create a trigger if sample is within the chopping time window of first_chop_sample to last_chop_sample  
  if (sample_nr < DEV_first_chop_sample[itq] || sample_nr > DEV_last_chop_sample[itq] ) return;

  // only create a trigger if sample is outside the T-method mask window
  if (tmask_prescale && DEV_TMask_window[itq] && sample_nr > DEV_mask_min[itq] && sample_nr < DEV_mask_max[itq] ) return; 

  while ( sample_nr < DEV_N_samples[itq] )
    {
      
      unsigned int idet;
      // leading edge threshold on individual segments
      if ( DEV_island_option[itq] == 2 || DEV_island_option[itq] == 4 ){
	double adc_cal = 0;
	for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	  {
	    ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];
	    // adc_cal = A_calib[idet]*(adc - auxdata->pedestal[idet]); // with calibration coefficient
	    adc_cal = (adc - auxdata->pedestal[idet]); // w/o calibration coefficient

	    if(adc_cal==0 || abs(adc_cal)>4096) continue; //check rational adc value
	   
	    if(DEV_threshold_sign[itq] && DEV_fill_type[itq]==2 && adc_cal>0) continue; //skip waveform headers for laser triggers (for positive pulses) 

            if ( DEV_useindividualthresholds[itq] ){
	      if ( DEV_thresholdpolarities[ DEV_rider_index[idet][itq] ][itq] && adc_cal > DEV_thresholdvalues[ DEV_rider_index[idet][itq] ][itq] ){
		auxdata->island_pattern[sample_nr] = 1;
                if ( DEV_island_option[itq] == 4 && sample_nr + TcycClockTicks < DEV_N_samples[itq] ) auxdata->island_pattern[ sample_nr + TcycClockTicks ] = 1;
		break;
	      }
	      if (!DEV_thresholdpolarities[ DEV_rider_index[idet][itq] ][itq] && adc_cal < DEV_thresholdvalues[ DEV_rider_index[idet][itq] ][itq] ){
		auxdata->island_pattern[sample_nr] = 1;
                if ( DEV_island_option[itq] == 4 && sample_nr + TcycClockTicks < DEV_N_samples[itq] ) auxdata->island_pattern[ sample_nr + TcycClockTicks ] = 1;
		break;
	      } // end individual thresholds
	    } else { 
	      if ( DEV_threshold_sign[itq] && adc_cal > DEV_threshold[itq] ){
		auxdata->island_pattern[sample_nr] = 1;
                if ( DEV_island_option[itq] == 4 && sample_nr + TcycClockTicks < DEV_N_samples[itq] ) auxdata->island_pattern[ sample_nr + TcycClockTicks ] = 1;
		break;
	      }
	      if ( !DEV_threshold_sign[itq] && adc_cal < DEV_threshold[itq] ){
		auxdata->island_pattern[sample_nr] = 1;
                if ( DEV_island_option[itq] == 4 && sample_nr + TcycClockTicks < DEV_N_samples[itq] ) auxdata->island_pattern[ sample_nr + TcycClockTicks ] = 1;
		break;
	      } 
	    }// end global threshols
	    
	  } // loop over detector
 
      } // island option	
      
      // pulseshape weighted threshold on individual segments
      if ( DEV_island_option[itq] == 3 ){
	
        // hard-coded pulseshape
	const int Nwgt = 7;
	double wgt[Nwgt] = {0.0625, 0.1250, 0.1875, 0.2500, 0.1875, 0.1250, 0.0625};
	int wgtlo = -3, wgthi = 3; 	  
	
	if ( (sample_nr > -wgtlo) && (sample_nr < (DEV_N_samples[itq] - wgthi)) ){  // waveform bookend
	  for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	    {
	      int firstsample = DEV_first_sample_index[idet][itq];

	      //double calconst = A_calib[idet]; // with calibration coefficient
	      double calconst = 1; // w/o calibration coefficient
	      double pedval = auxdata->pedestal[idet];
	      
	      double adcwgt = 0.0;
              int iwgt = 0;
	      for (iwgt = wgtlo; iwgt <= wgthi; iwgt++){
	        ADC_TYPE adc =  gpu_idata[firstsample + sample_nr - iwgt];
		adcwgt += wgt[iwgt - wgtlo] * calconst * (adc - pedval);
	      }

	      if ( DEV_threshold_sign[itq] && adcwgt > DEV_threshold[itq] ){  // positive-going trigger
		auxdata->island_pattern[sample_nr] = 1;
		break;
	      }
	      if ( !DEV_threshold_sign[itq] && adcwgt < DEV_threshold[itq] ){ // negative-going trigger
		auxdata->island_pattern[sample_nr] = 1;
		break;
	      }

	    } // loop over detectors
	} // waveform bookend
      } // island option

      sample_nr += blockDim.x * gridDim.x;
    }
}

__global__
void kernel_extend_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
 
  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples[itq] )
    {     
      int is_BOI = 0; // beginning of an island
      int is_EOI = 0; // end of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr] > 0 )
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr-1] == 0 )
	      is_BOI=1;
	  
	   // check EOI condition
	  if ( sample_nr == (DEV_N_samples[itq]-1) )
	    is_EOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr+1] == 0 )
	      is_EOI=1;
	}
	
      if ( is_BOI )
	{
	  // This is a beginning of an island
	  // extend the island for N_presamples

	  //printf("kernel_extend_island::: BOI, sample_nr %i, array index \n",  sample_nr, sample_nr); //debug

	  int i1 = sample_nr - DEV_N_presamples[itq];
	  if ( i1 < 0 ) i1 = 0;
	  int k;
	  for (k=i1; k<sample_nr; k++)
	    {
	      atomicAdd( &(auxdata->island_pattern[k]), 1);		  
	    }
	}

      // check the "End Of Island" condition
      if ( is_EOI )
	{
	  // This is an end of an island
	  // extend the island for N_postsamples
	  int i2 = sample_nr + DEV_N_postsamples[itq];
	  if ( i2 >= DEV_N_samples[itq] ) i2 = DEV_N_samples[itq]-1;
	  int k;
	  for (k=i2; k>sample_nr; k--)
	    {
	      atomicAdd( &(auxdata->island_pattern[k]), 1);		  
	    }
	}

      sample_nr += blockDim.x * gridDim.x;
    }
}

__global__
void kernel_find_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  //find new islands (some of the original islands could have merged together) 
  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples[itq] )
    {

      int is_BOI = 0; // beginning of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr] > 0 )
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr-1] == 0 )
	      is_BOI=1;
	}


      if ( is_BOI )
	{
	  // This is a beginning of an island


	  //printf("kernel_find_island::: BOI, sample_nr %i, array index \n",  sample_nr, sample_nr); //debug
	  
	  // island number
	  int island_nr = atomicAdd( &(outdata->n_islands), 1); 
	  auxdata->island_info[island_nr].time = sample_nr;
	  
	  // determine the length of the island
	  int i;
	  int island_nr_aux = island_nr + 1;
	  for (i=sample_nr; i<DEV_N_samples[itq]; i++)
	    {
	      if ( auxdata->island_pattern[i] == 0 )
		{
		  break;
		}
	      else
		{
		  auxdata->island_pattern[i] = island_nr_aux;
		}
	    }
	  int island_len = i - sample_nr;
	  // record the length into first bin
	  auxdata->island_info[island_nr].length = island_len;

          // the ugly 4 accounts for two 32bit islander header words - the island time and island length
	  int offset = atomicAdd( &(outdata->island_offset), 4 + DEV_N_detectors[itq]*island_len);
	  auxdata->island_info[island_nr].offset = offset;
	  
          memcpy( &outdata->islands[offset], &sample_nr, sizeof(int) ); 
          memcpy( &outdata->islands[offset+2], &island_len, sizeof(int) ); 	  
	}

      sample_nr += blockDim.x * gridDim.x; 
    }
}

__global__
void kernel_make_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  //find new islands (some of the old island could have merged   
  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples[itq] )
    {
      int island_nr = auxdata->island_pattern[sample_nr];
      if ( island_nr > 0 )
	{
	  island_nr--;
	  // +4 is needed to skip 32-bit time and length words in 16-bit array
	  int island_offset    = auxdata->island_info[island_nr].offset + 4;  
	  int island_sample_nr = sample_nr - auxdata->island_info[island_nr].time;
	  int island_length    = auxdata->island_info[island_nr].length;
	  int idet;
	  for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	    {
	      int i = island_offset + idet*island_length + island_sample_nr;
	      ADC_TYPE adc=gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];
	      outdata->islands[i] = adc;
	    }
	}
      sample_nr += blockDim.x * gridDim.x; 
    }
}

/** 
 * Find the triggers in waveforms
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_xtaltrigger(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq, bool tmask_prescale)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id, block id, .. to define the sample index
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  const unsigned int num_threads = blockDim.x;

  int sample_nr = tid + bid*num_threads;

  // only create a trigger if sample is within the chopping time window of first_chop_sample to last_chop_sample  
  if (sample_nr < DEV_first_chop_sample[itq] || sample_nr > DEV_last_chop_sample[itq] ) return;

  // only create a trigger if sample is outside the T-method mask window
  if (tmask_prescale && DEV_TMask_window[itq] && sample_nr > DEV_mask_min[itq] && sample_nr < DEV_mask_max[itq] ) return; 

  while ( sample_nr < DEV_N_samples[itq] )
    {
      
      unsigned int idet;
      
      // leading edge threshold on individual segments
      if ( DEV_island_option[itq] == 2 ){
	double adc_cal = 0;
	for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	  {
	    ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];
	    // adc_cal = A_calib[idet]*(adc - auxdata->pedestal[idet]); // with calibration coefficient
	    adc_cal = (adc - auxdata->pedestal[idet]); // w/o calibration coefficient

            if ( DEV_useindividualthresholds[itq] ){
	      if ( DEV_thresholdpolarities[ DEV_rider_index[idet][itq] ][itq] && adc_cal > DEV_thresholdvalues[ DEV_rider_index[idet][itq] ][itq] ){
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      }
	      if (!DEV_thresholdpolarities[ DEV_rider_index[idet][itq] ][itq] && adc_cal < DEV_thresholdvalues[ DEV_rider_index[idet][itq] ][itq] ){
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      } // end individual thresholds
	    } else { 
	      if ( DEV_threshold_sign[itq] && adc_cal > DEV_threshold[itq] ){
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		//printf("kernel_wf_trigger::: sample_nr %i , detector_nr  %i, array index %i\n",  sample_nr, idet, sample_nr + idet*DEV_N_samples[itq]); //debug
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      }
	      if ( !DEV_threshold_sign[itq] && adc_cal < DEV_threshold[itq] ){
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		//printf("kernel_wf_trigger::: sample_nr %i , detector_nr  %i, array index \n",  sample_nr, idet, sample_nr + idet*DEV_N_samples[itq]); //debug
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      } 
	    }// end global threshols
	    
	  } // loop over detector
 
      } // island option	
      
      // pulseshape weighted threshold on individual segments
      if ( DEV_island_option[itq] == 3 ){
	
        // hard-coded pulseshape
	const int Nwgt = 7;
	double wgt[Nwgt] = {0.0625, 0.1250, 0.1875, 0.2500, 0.1875, 0.1250, 0.0625};
	int wgtlo = -3, wgthi = 3; 	  
	
	if ( (sample_nr > -wgtlo) && (sample_nr < (DEV_N_samples[itq] - wgthi)) ){  // waveform bookend
	  for (idet=0; idet<DEV_N_detectors[itq]; idet++)
	    {
	      int firstsample = DEV_first_sample_index[idet][itq];

	      //double calconst = A_calib[idet]; // with calibration coefficient
	      double calconst = 1; // w/o calibration coefficient
	      double pedval = auxdata->pedestal[idet];
	      
	      double adcwgt = 0.0;
              int iwgt = 0;
	      for (iwgt = wgtlo; iwgt <= wgthi; iwgt++){
	        ADC_TYPE adc =  gpu_idata[firstsample + sample_nr - iwgt];
		adcwgt += wgt[iwgt - wgtlo] * calconst * (adc - pedval);
	      }

	      if ( DEV_threshold_sign[itq] && adcwgt > DEV_threshold[itq] ){  // positive-going trigger
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      }
	      if ( !DEV_threshold_sign[itq] && adcwgt < DEV_threshold[itq] ){ // negative-going trigger
		auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] = 1;
		continue; // need to continue not break when filling a samples * detectors island_pattern array
	      }

	    } // loop over detectors
	} // waveform bookend
      } // island option

      sample_nr += blockDim.x * gridDim.x;
    }
}


__global__
void kernel_extend_xtalislands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
 
  int sampletimesdetector_nr = tid + bid*num_threads; 
  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ){

    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];  // for given detector
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];  // detector identifier

  while ( sample_nr < DEV_N_samples[itq] )
    {     
      int is_BOI = 0; // beginning of an island
      int is_EOI = 0; // end of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] > 0 )
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr - 1 + idet*DEV_N_samples[itq]] == 0 )
	      is_BOI=1;
	  
	   // check EOI condition
	  if ( sample_nr == (DEV_N_samples[itq]-1) )
	    is_EOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr + 1 + idet*DEV_N_samples[itq]] == 0 )
	      is_EOI=1;
	}
	
      if ( is_BOI )
	{
	  // This is a beginning of an island
	  // extend the island for N_presamples

	  //printf("kernel_extend_xtalisland::: BOI, sample_nr %i , detector_nr  %i, array index \n",  sample_nr, idet, sample_nr + idet*DEV_N_samples[itq]); //debug

	  int i1 = sample_nr - DEV_N_presamples[itq];
	  if ( i1 < 0 ) i1 = 0;
	  int k;
	  for (k=i1; k<sample_nr; k++)
	    {
	      //atomicAdd( &(auxdata->island_pattern[k + idet*DEV_N_samples[itq]]), 1);		  
	      auxdata->island_pattern[k + idet*DEV_N_samples[itq]] = 1; // no need for atomic add with nsamples*ndetectors array		  
	    }

	}

      // check the "End Of Island" condition
      if ( is_EOI )
	{
	  // This is an end of an island
	  // extend the island for N_postsamples

	  //printf("kernel_extend_island::: EOI, sample_nr %i , detector_nr  %i, array index \n",  sample_nr, idet, sample_nr + idet*DEV_N_samples[itq]); //debug

	  int i2 = sample_nr + DEV_N_postsamples[itq];
	  if ( i2 >= DEV_N_samples[itq] ) i2 = DEV_N_samples[itq]-1;
	  int k;
	  for (k=i2; k>sample_nr; k--)
	    {
	      //atomicAdd( &(auxdata->island_pattern[k + idet*DEV_N_samples[itq]]), 1);		  
	      auxdata->island_pattern[k + idet*DEV_N_samples[itq]] = 1; // no need for atomic add with nsamples*ndetectors array		  
	    }
	}

      sample_nr += blockDim.x * gridDim.x;
    }
  }
}

/** 
 * Find the border samples around trigger samples
 * 
 * border samples are identified as = 2 in island_pattern[]
 * trigger samples are identified as = 1 in island_pattern[]
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_border_xtalislands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq) {

  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
 
  int sampletimesdetector_nr = tid + bid*num_threads; 
  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ){
  
    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];  // for given detector
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];  // detector identifier
  
  while ( sample_nr < DEV_N_samples[itq] ){ // what about first_chop, last_chop?
    
      if ( auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]] == 1 ){ // look for trigger sample identifier (=1)
	
	int iX = idet % DEV_N_segments_x[itq]; // x-coordinate of trigger xtal
	int iY = idet / DEV_N_segments_x[itq]; // y-coordinate of trigger xtal

	//printf("kernel_border_island:  NX %i, NY %i, sample_nr %i , detector_nr  %i, iX %i, iY %i, array index %i\n", DEV_N_segments_x[itq], DEV_N_segments_y[itq], sample_nr, idet, iX, iY, sample_nr + idet*DEV_N_samples[itq]); //debug

	for (int ixborder = iX-1; ixborder <= iX+1; ixborder++) {
	  for (int iyborder = iY-1; iyborder <= iY+1; iyborder++) {

	    int jdet = ixborder + iyborder*DEV_N_segments_x[itq]; // border xtal index
	    //printf("kernel_border_xtalisland: iX %i, iY %i, ixborder %i, iyborder %i, idet %i, jdet %i, DEV_N_segments_x[itq] %i\n", iX, iY, ixborder, iyborder, idet, jdet, DEV_N_segments_x[itq]);
	    
	    if ( iX == ixborder && iY == iyborder ) continue; // skip trigger sample
	    if ( ixborder < 0 || ixborder >= DEV_N_segments_x[itq] ) continue; // out of range or detector array 
	    if ( iyborder < 0 || iyborder >= DEV_N_segments_y[itq] ) continue; // out of range or detector array 
	    
            if ( auxdata->island_pattern[sample_nr + jdet*DEV_N_samples[itq]] == 0 )
	       auxdata->island_pattern[sample_nr + jdet*DEV_N_samples[itq]] = 2; // write  border sample  identifier (=2)   

	  } // end y-border loop
	} // end x-border loop	
      } // end found trigger sample

      sample_nr += blockDim.x * gridDim.x;
    } // end while sample 
  } // if samples*detectors
}

__global__
void kernel_find_xtalislands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  int sampletimesdetector_nr = tid + bid*num_threads; 
  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ){

    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];  // for given detector
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];  // detector identifier

  //find new islands (some of the original islands could have merged together) 
  while ( sample_nr < DEV_N_samples[itq] )
    {

      int is_BOI = 0; // beginning of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr  + idet*DEV_N_samples[itq]] > 0 ) // > 0 recognizes trigger and border samples
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr - 1  + idet*DEV_N_samples[itq]] == 0 )
	      is_BOI=1;
	}


      if ( is_BOI )
	{
	  // This is a beginning of an island

	  //printf("kernel_find_xtalisland::: BOI, sample_nr %i , detector_nr  %i, array index %i, island pattern %i\n",  
	  //	 sample_nr, idet, sample_nr + idet*DEV_N_samples[itq], auxdata->island_pattern[sample_nr - 1  + idet*DEV_N_samples[itq]]); //debug
	  
	  // island number
	  int island_nr = atomicAdd( &(outdata->n_islands), 1); 
	  auxdata->island_info[island_nr].time = sample_nr;
	  auxdata->island_info[island_nr].detector = idet;
	  
	  // determine the length of the island
	  int i;
	  int island_nr_aux = island_nr + 1;
	  for (i=sample_nr; i<DEV_N_samples[itq]; i++)
	    {
	      if ( auxdata->island_pattern[i + idet*DEV_N_samples[itq]] == 0 )
		{
		  break;
		}
	      else
		{
		  auxdata->island_pattern[i + idet*DEV_N_samples[itq]] = island_nr_aux; // writes island number to island_pattern[], used by kernel_make island
		}
	    }
	  int island_len = i - sample_nr;
	  // record the length into first bin
	  auxdata->island_info[island_nr].length = island_len;

          // the ugly 4 accounts for two 32bit islander header words - the island time and island length
	  // below changed ndet*island_length to island_length when writing each xtals as individual island
          // changed ugly 4 to ugly 6 as added the writing of detector index as additional header word for each island
	  int offset = atomicAdd( &(outdata->island_offset), 6 + island_len); 
	  auxdata->island_info[island_nr].offset = offset;

          memcpy( &outdata->islands[offset], &sample_nr, sizeof(int) ); 
          memcpy( &outdata->islands[offset+2], &island_len, sizeof(int) ); 	  
          memcpy( &outdata->islands[offset+4], &idet, sizeof(int) );
	}

      sample_nr += blockDim.x * gridDim.x; 
    }
  }
}

__global__
void kernel_make_xtalislands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  //find new islands (some of the old island could have merged   
  int sampletimesdetector_nr = tid + bid*num_threads; 
  if ( sampletimesdetector_nr < DEV_N_samples[itq]*DEV_N_detectors[itq] ){
    
    int sample_nr = sampletimesdetector_nr % DEV_N_samples[itq];  // for given detector
    int idet = sampletimesdetector_nr / DEV_N_samples[itq];  // detector identifier
    
    while ( sample_nr < DEV_N_samples[itq] )
      {
	int island_nr = auxdata->island_pattern[sample_nr + idet*DEV_N_samples[itq]];
	if ( island_nr > 0 )
	  {
	    island_nr--;
	    // +4 is needed to skip 32-bit time and length words in 16-bit array
	    // changed +4 to +6 as extra header word for dtector index
	    int island_offset    = auxdata->island_info[island_nr].offset + 6;  
	    int island_sample_nr = sample_nr - auxdata->island_info[island_nr].time;
	    //int island_length    = auxdata->island_info[island_nr].length;
	    int idet = auxdata->island_info[island_nr].detector;
	    
	    //printf("kernel_make_island::: island_nr %i, sample_nr %i, islandsample_nr %i, detector_nr  %i\n",  island_nr, sample_nr, island_sample_nr, idet ); //debug
	
            int i = island_offset + island_sample_nr;    
	    ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq] + sample_nr];
	    outdata->islands[i] = adc;
	  }
	sample_nr += blockDim.x * gridDim.x; 
      }
  }
}

/** 
 * Evaluate pedestals
 * option 0 - global pedestal from ODB
 * option 1 - fill-by-fill calculation
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_make_pedestals(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
 
  int idet = tid;  // detector number 
   
  // pedestal option 1, fill-by-fill calculation of pedestal
  if (DEV_pedestal_option[itq] == 1) {

    const int nsamples = 100; // number of samples for averaging
    double adc_mean = 0; // mean value of adc samples
    int i;
        // calculation on pedestal using first nsamples of fill
    for (i=0; i<nsamples; i++)
      {
	ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet][itq]+i];
	adc_mean += adc;
      }
    
    // sum -> average and store in pedestal array
    adc_mean /= nsamples;
    auxdata->pedestal[idet] = adc_mean;

    //printf("kernel_make_pedestals: idet %i, itq %i, DEV_first_sample_index[idet][itq] %i gpu_idata[DEV_first_sample_index[idet][itq]] %i, pedestal %f\n",
    //	   idet, itq, DEV_first_sample_index[idet][itq], gpu_idata[DEV_first_sample_index[idet][itq]], adc_mean);

  }

  // pedestal option 0, global pedestal from ODB 
  if (DEV_pedestal_option[itq] == 0) {

    auxdata->pedestal[idet] = DEV_global_pedestal[itq];

  }
  
}

__global__
void kernel_calc_ctag( void* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int sample_nr = bid*num_threads + tid;

  // example counting of calorimeter hits
  //int threshold_high = DEV_ctag_threshold;;

  if(sample_nr>DEV_ctag_time_cut[itq] && sample_nr<DEV_N_samples[itq] && auxdata->wf_sum[sample_nr]<DEV_ctag_threshold[itq]) {
    if( auxdata->wf_sum[sample_nr]<auxdata->wf_sum[sample_nr+1] && auxdata->wf_sum[sample_nr]<auxdata->wf_sum[sample_nr-1]) { //ONLY COUNT THE PEAK
      
      atomicAdd(&(outdata->CTAG),1); // adc>2GeV && t>50us

    } 
  }
}
  
__global__
void kernel_decimate_sum( void* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  
  int sample_nr_1 = (tid + bid*num_threads)*DEV_decimation[itq];
  int sample_nr_2 = sample_nr_1 +  DEV_decimation[itq];

  int i;
  double adc_sum = 0;
  for (i=sample_nr_1; i<sample_nr_2; i++)
    {
      adc_sum += auxdata->wf_sum[i];
      //break; // makes the sum just first sample of DEV_decimation samples
    }
  
  // append the sum to the end of the output data
  // the ugly "6" accounts for 6x16-bit header words for data size (size 2), 
  // island number (size 1), detector number (size 1), CTAG (size 2)
  // the "ugly" +1 pads an odd number of 16-bit T-method data words to the
  // next 32-bit word boundary. 

  //old 16-bit fill-by-fill, decimated histo
  //int16_t *data = (int16_t*)(outdata) + (6 + outdata->island_offset); 
  //data[ sample_nr_1 / DEV_decimation[itq] ] = (int16_t) ( adc_sum ); 
 
  //new 32-bit fill-by-fill, decimated histo
  int32_t *data;
  if (outdata->island_offset % 2 == 0)  data = (int32_t*)(outdata) + (6 + outdata->island_offset)/2;
  else data = (int32_t*)(outdata) + (6 + outdata->island_offset + 1)/2;

  data[ sample_nr_1 / DEV_decimation[itq] ] = (int32_t) ( adc_sum );
}

__global__
void kernel_pedestal_store( void* gpu_odata, int itq)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id
  const unsigned int tid = threadIdx.x;

  // get detector number
  int idet = tid; 

  // append the pedestal to the end of the output data via
  // pointer to outdata + island header size + island data size + Q-method data size
  // the ugly "6" accounts for 6x16-bit header words for data size (size 2), 
  // island number (size 1), detector number (size 1), CTAG (size 2)
  // the "ugly" +1 pads an odd number of 16-bit T-method data words to the
  // next 32-bit word boundary. 
  
  //old 16-bit fill-by-fill, decimated histo + pedestal data
  //int16_t *data = (int16_t*)(outdata) + (6 + outdata->island_offset) + DEV_N_samples[itq] / DEV_decimation[itq];
  //old 16-bit fill-by-fill, decimated histo + pedestal data
  float *data;
  if (outdata->island_offset % 2 == 0)  data = (float*)(outdata) + (6 + outdata->island_offset)/2 + (DEV_N_samples[itq]/DEV_decimation[itq]);
  else data = (float*)(outdata) + (6 + outdata->island_offset + 1)/2 + (DEV_N_samples[itq]/DEV_decimation[itq]);
  
  // writeout pedestal value
  data[idet] = (float) auxdata->pedestal[idet];
  //printf("kernel_pedestal_store: idet %i, itq %i, ped %f\n",  idet, itq, data[idet]);
}

// ATF, read in binary pulse template file
void read_template(const std::string& fname, pulseTemplate& templ,
                   phaseMap& pmap) {
  std::ifstream ifile(fname.c_str(), std::ifstream::binary);

  unsigned int samplesperfit;
  ifile.read((char*)&samplesperfit, sizeof(samplesperfit));
  assert(samplesperfit == SAMPLESPERFIT);

  unsigned int pointspersample;
  ifile.read((char*)&pointspersample, sizeof(pointspersample));
  assert(pointspersample == POINTSPERSAMPLE);

  ifile.read((char*)templ.table, NPOINTSTEMPLATE * sizeof(float));

  unsigned int npointsphasemap;
  ifile.read((char*)&npointsphasemap, sizeof(npointsphasemap));
  assert(npointsphasemap == NPOINTSPHASEMAP);

  ifile.read((char*)pmap.table, NPOINTSPHASEMAP * sizeof(float));

  ifile.read((char*)&pmap.timeOffset, sizeof(pmap.timeOffset));
}

/*
 * cuda_g2_bor_kernel() called at begin-of-run to copy the paramters for data processing from 
 * the host to device.
 *
 * gpu_idata input data of continuous samples from gpu_thread to GPU
 * cpu_odata output data of T-method, Q-method to gpu_thread from GPU
 *
 */

void cuda_g2_bor_kernel(){

  hipError_t cudaCopyStatus;
  int rider_mod_max = 12, rider_chn_max = 5;

  dbprintf("cuda_g2_bor_kernel(), TQMETHOD_MAX = %d\n", TQMETHOD_MAX);

  // needed for accessing the CaloMap to make the array of first samples
  //amc13_ODB_get(); Already called in the top-level begin_of_run function
  
  // prepare processing parameters from ODB-mapped c-structure, etc
  for (int i = 0; i < TQMETHOD_MAX; i++){	       

    HOST_N_samples[i] = tq_parameters_odb[i].gpu_waveform_length;
    HOST_first_chop_sample[i] = tq_parameters_odb[i].gpu_waveform_firstsample;
    HOST_last_chop_sample[i] = tq_parameters_odb[i].gpu_waveform_lastsample;
    HOST_N_segments_x[i] = tq_parameters_odb[i].gpu_n_segments_x;
    HOST_N_segments_y[i] = tq_parameters_odb[i].gpu_n_segments_y;
    HOST_N_presamples[i] = tq_parameters_odb[i].gpu_island_presamples;
    HOST_N_postsamples[i] = tq_parameters_odb[i].gpu_island_postsamples;
    HOST_decimation[i] = tq_parameters_odb[i].calosum_decimation_factor;
    HOST_island_option[i] = tq_parameters_odb[i].island_option;
    HOST_threshold[i] = tq_parameters_odb[i].T_threshold;
    HOST_threshold_sign[i] = tq_parameters_odb[i].T_threshold_sign;
    HOST_useindividualthresholds[i] = tq_parameters_odb[i].use_channel_thresholds;
    HOST_pedestal_option[i] = tq_parameters_odb[i].pedestal_option;
    HOST_global_pedestal[i] = tq_parameters_odb[i].global_pedestal;
    HOST_hpedsubtract[i] = tq_parameters_odb[i].subtract_ped;
    HOST_hdecimation[i] = tq_parameters_odb[i].time_divide_hist;
    HOST_hfirstsample[i] = tq_parameters_odb[i].first_sample_in_hist;
    HOST_hlastsample[i] = tq_parameters_odb[i].last_sample_in_hist;
    HOST_hrebinintervals[i] = tq_parameters_odb[i].rebin_intervals_in_hist;
    HOST_hrebinincrement[i] = tq_parameters_odb[i].rebin_increment_in_hist;
    HOST_hoffset[i] = GPU_Data_Buffer[0].gpu_data_his_offset[i] / sizeof(int32_t);     // convert size from bytes to 32-bit words of histogram array
    HOST_hsize[i] = GPU_Data_Buffer[0].gpu_data_his_size[i] / sizeof(int32_t);
    HOST_nfitislands[i] = tq_parameters_odb[i].fit_islands;
    HOST_minfittime[i] = tq_parameters_odb[i].min_fit_time;
    HOST_fit_threshold[i] = tq_parameters_odb[i].fit_threshold;
    HOST_ctag_threshold[i] = tq_parameters_odb[i].CTAG_threshold;
    HOST_ctag_time_cut[i] = tq_parameters_odb[i].CTAG_time_cut;
    HOST_TMask_window[i] = tq_parameters_odb[i].TMask_window;
    HOST_mask_min[i] = tq_parameters_odb[i].mask_min;
    HOST_mask_max[i] = tq_parameters_odb[i].mask_max;
    HOST_mask_prescale[i] = tq_parameters_odb[i].mask_prescale;
    HOST_save_full_calo[i] = tq_parameters_odb[i].save_full_calo;
    HOST_save_xtal_border[i] = tq_parameters_odb[i].save_xtal_border;
    HOST_fit_prescale_factor[i] = tq_parameters_odb[i].fit_prescale_factor;
    HOST_fill_type[i] = tq_parameters_odb[i].fill_type;

    // set sequence index when filling histograms by sequence number using GPUmuonfillnumber (update to fill index from AMC13 header) 
    int fill_seq = 8;
    if  ( tq_parameters_odb[i].separate_sequence_hist ) {
      HOST_fill_seq[i] = fill_seq;
    } else {
      HOST_fill_seq[i] = 1;
    }

    for (int im = 0; im < rider_mod_max; im++){
      for (int ic = 0; ic < rider_chn_max; ic++){
	int index = ic + im*rider_chn_max;
	HOST_thresholdvalues[index][i] = rider_map_to_calo_odb[im][ic][i].value;
	HOST_thresholdpolarities[index][i] = rider_map_to_calo_odb[im][ic][i].polarity;
        dbprintf("im %i, ic %i, index %i, TQ %i, HOST_thresholdvalues %i, HOST_thresholdpolarities %i\n", 
	         im, ic, index, i, HOST_thresholdvalues[index][i], HOST_thresholdpolarities[index][i]);
      } // loop over rider channels
    } // loop over rider modules
  } // loop of TQ methods
  
  hipError_t dev_set = hipSetDevice( amc13_settings_odb.gpu_dev_id );
  if ( dev_set != hipSuccess )
    {
      printf("ERROR: (gpu_bor) acquiring CUDA device\n");
    }
   
  // calculate the array index of first sample of each calo segment accounting for
  // (i) the Rider module/channel/waveform header/trailer words within AMC payload
  // (ii) the mapping between the calorimeter segments and the rider modules / channels

  int index;
  int ix, iy, im, ic, itq, idet;
 
  for (itq = 0; itq < TQMETHOD_MAX; itq++){

    dbprintf("cuda_g2_bor_kernel(), make map itq = %d\n", itq);

    index = 0; // reset first sample insex for each TQ method
    idet = 0; // enabled detector counter "enabled detector" type map

    for(im=0; im<rider_mod_max; im++){
      if (amc13_rider_odb[im].board.rider_enabled) {
      
        // account for Rider module header words
#ifdef USE_RIDER_FORMAT
        index += NRMH_WORDS;
#endif

	for(ic=0; ic<rider_chn_max; ic++){
	if (amc13_rider_odb[im].channel[ic].enabled) {
	  
      // account for Rider channel header words
#ifdef USE_RIDER_FORMAT
          index += NRCH_WORDS;
#endif

          if (tq_parameters_odb[itq].TQ_maptype == 0){
	    ix = rider_map_to_calo_odb[im][ic][itq].x_segment;
	    iy = rider_map_to_calo_odb[im][ic][itq].y_segment;
	    dbprintf("cuda_g2_bor_kernel(): im = %i, ic = %i, ix = %i, iy = %i\n",im,ic,ix,iy);
	  
	    if (ix >= 1 && ix <= HOST_N_segments_x[itq] && iy >= 1 && iy <= HOST_N_segments_y[itq] && rider_map_to_calo_odb[im][ic][itq].enabled) {
	      HOST_first_sample_index[ (ix-1) + (iy-1)*HOST_N_segments_x[itq] ][itq] = index;
	      HOST_rider_index[ (ix-1) + (iy-1)*HOST_N_segments_x[itq] ][itq] = ic + im*rider_chn_max;
	      dbprintf("calo segment x,y %i, %i  first_sample_index %i\n", ix, iy, HOST_first_sample_index[ (ix-1) + (iy-1)*HOST_N_segments_x[itq] ][itq] );
	      idet++;
	    }
	  }
          if (tq_parameters_odb[itq].TQ_maptype == 1) {
	    if ( rider_map_to_calo_odb[im][ic][itq].enabled ) {
	      HOST_first_sample_index[idet][itq] = index;
	      dbprintf("enabled detector %i  first_sample_index %i\n", idet, HOST_first_sample_index[idet][itq] );
              idet++;
	  }
	}
	
          // assume all channels have wavefrom of length gpu_waveform_length
          // calo processing is only sensible for equal length waveforms from all segments
	  // index += tq_parameters_odb[itq].gpu_waveform_length;
  
          // fix to allow different rider modules to have different fill lengths
          index += amc13_rider_odb[im].board.trig1__wvfm_length;

      // account for Rider channel trailer words
#ifdef USE_RIDER_FORMAT
	  index += NRCT_WORDS;
#endif
  	  } // if for enabled rider channels
        } // for loop on rider channels  

      // account for Rider module trailer words
#ifdef USE_RIDER_FORMAT
       index += NRMT_WORDS;
#endif
        } // if for enabled rider modules
     } // for loop on rider channels

    HOST_N_detectors[itq] = idet;
    if ( tq_parameters_odb[itq].TQ_on || tq_parameters_odb[itq].store_hist ){
      if ( HOST_N_detectors[itq] != HOST_N_segments_x[itq]*HOST_N_segments_y[itq] ){
	printf("ERROR: enabled channels and array size do not match!");
	cm_msg(MERROR, __FILE__, "enabled channels and array size do not match! TQ index %i, detector number %i, array dimensions %ix%i = %i\n", 
	       itq, HOST_N_detectors[itq], HOST_N_segments_x[itq], HOST_N_segments_y[itq], HOST_N_segments_x[itq]*HOST_N_segments_y[itq] );
      }
    }
 
 } // for loop on TQMETHODS

  // copy gpu analysis parameters to device

  // length of waveform
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_samples), HOST_N_samples, sizeof(HOST_N_samples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d \n", sizeof(HOST_N_samples));
    }
  dbprintf("hipMemcpyToSymbol of number of samples[0] %i, size %i, status %i \n", HOST_N_samples[0], sizeof(HOST_N_samples), (int)cudaCopyStatus );

  // first sample for chopping
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_first_chop_sample), HOST_first_chop_sample, sizeof(HOST_first_chop_sample), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of first_chop_sample FAIL, bytes %d \n", sizeof(HOST_first_chop_sample));
    }
  dbprintf("hipMemcpyToSymbol of first chop sample[0] %i, size %i, status %i \n", HOST_first_chop_sample[0], sizeof(HOST_first_chop_sample), (int)cudaCopyStatus );

  // last sample for chopping
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_last_chop_sample), HOST_last_chop_sample, sizeof(HOST_last_chop_sample), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of last_chop_sample FAIL, bytes %d \n", sizeof(HOST_last_chop_sample));
    }
  dbprintf("hipMemcpyToSymbol of last chop sample[0] %i, size %i, status %i \n", HOST_last_chop_sample[0], sizeof(HOST_last_chop_sample), (int)cudaCopyStatus );

  // number of detectors
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_detectors), HOST_N_detectors, sizeof(HOST_N_detectors), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d \n", sizeof(HOST_N_detectors));
    }
  dbprintf("hipMemcpyToSymbol of number of detectors[0] %i, size %i, status %i \n", HOST_N_detectors[0], sizeof(HOST_N_detectors), (int)cudaCopyStatus );

  // number of segments_x
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_segments_x), HOST_N_segments_x, sizeof(HOST_N_segments_x), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d \n", sizeof(HOST_N_segments_x));
    }
  dbprintf("hipMemcpyToSymbol of number of segments_x[0] %i, size %i, status %i \n", HOST_N_segments_x[0], sizeof(HOST_N_segments_x), (int)cudaCopyStatus );

  // number of segments_y
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_segments_y), HOST_N_segments_y, sizeof(HOST_N_segments_y), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d \n", sizeof(HOST_N_segments_y));
    }
  dbprintf("hipMemcpyToSymbol of number of segments_y[0] %i, size %i, status %i \n", HOST_N_segments_y[0], sizeof(HOST_N_segments_y), (int)cudaCopyStatus );

  // N_detector array for first samples of detector segments
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_first_sample_index), HOST_first_sample_index, sizeof(HOST_first_sample_index), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of first sample indices FAIL, bytes %d \n", sizeof(HOST_first_sample_index));
    }
  dbprintf("hipMemcpyToSymbol of first sample indices[0][0] %i, size %i, status %i \n", HOST_first_sample_index[0][0], sizeof(HOST_first_sample_index), (int)cudaCopyStatus );

  // N_detector array for rider index
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_rider_index), HOST_rider_index, sizeof(HOST_rider_index), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of rider-detector map indices FAIL, bytes %d \n", sizeof(HOST_rider_index));
    }
  dbprintf("hipMemcpyToSymbol of rider-detector map indices[0][0] %i, size %i, status %i \n", HOST_rider_index[0][0], sizeof(HOST_rider_index), (int)cudaCopyStatus );

  // number of pre-samples and post-samples for T-method
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_presamples), HOST_N_presamples, sizeof(HOST_N_presamples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_presamples FAIL, bytes %d \n", sizeof(HOST_N_presamples));
    }
  dbprintf("hipMemcpyToSymbol of number of island pre-samples[0] %i, size %i, status %i \n", HOST_N_presamples[0], sizeof(HOST_N_presamples), (int)cudaCopyStatus );
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_postsamples), HOST_N_postsamples, sizeof(HOST_N_postsamples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_postsamples FAIL, bytes %d \n", sizeof(HOST_N_postsamples));
    }
  dbprintf("hipMemcpyToSymbol of number of island post-samples[0] %i, size %i, status %i \n", HOST_N_postsamples[0], sizeof(HOST_N_postsamples), (int)cudaCopyStatus );

  // T-method island option and threshold parameters
 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_island_option), HOST_island_option, sizeof(HOST_island_option), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of island_options FAIL, bytes %d \n", sizeof(HOST_island_option));
    }	
   dbprintf("hipMemcpyToSymbol of island_option[0] %i, size %i, status %i \n", HOST_island_option[0], sizeof(HOST_island_option), (int)cudaCopyStatus );
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_threshold), HOST_threshold, sizeof(HOST_threshold), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of thresholds FAIL, bytes %d \n", sizeof(HOST_threshold));
    }	
   dbprintf("hipMemcpyToSymbol of threshold[0] %i, size %i, status %i \n", HOST_threshold[0], sizeof(HOST_threshold), (int)cudaCopyStatus );
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_threshold_sign), HOST_threshold_sign, sizeof(HOST_threshold_sign), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of bool threshold sign FAIL, bytes %d \n", sizeof(HOST_threshold_sign));
    }	
   dbprintf("hipMemcpyToSymbol of bool threshold sign[0] %i, size %i, status %i \n", HOST_threshold_sign[0], sizeof(HOST_threshold_sign), (int)cudaCopyStatus );

  // T-method pedestal option and pedestal parameters
 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_pedestal_option), HOST_pedestal_option, sizeof(HOST_pedestal_option), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of pedestal_options FAIL, bytes %d \n", sizeof(HOST_pedestal_option));
    }	
   dbprintf("hipMemcpyToSymbol of pedestal_option[0] %i, size %i, status %i \n", HOST_pedestal_option[0], sizeof(HOST_pedestal_option), (int)cudaCopyStatus );
 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_global_pedestal), HOST_global_pedestal, sizeof(HOST_global_pedestal), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of global_pedestals FAIL, bytes %d \n", sizeof(HOST_global_pedestal));
    }	
   dbprintf("hipMemcpyToSymbol of global_pedestal[0] %i, size %i, status %i \n", HOST_global_pedestal[0], sizeof(HOST_global_pedestal), (int)cudaCopyStatus );

  // pedestal subtraction flag for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hpedsubtract), HOST_hpedsubtract, sizeof(HOST_hpedsubtract), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of bool threshold sign FAIL, bytes %d \n", sizeof(HOST_hpedsubtract));
    }	
   dbprintf("hipMemcpyToSymbol of bool fill-summed pedestal subtraction[0] %i, size %i, status %i \n", HOST_hpedsubtract[0], sizeof(HOST_hpedsubtract), (int)cudaCopyStatus );


  // first sample for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hfirstsample), HOST_hfirstsample, sizeof(HOST_hfirstsample), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histo first sample FAIL, bytes %d \n", sizeof(HOST_hfirstsample));
    }
  dbprintf("hipMemcpyToSymbol of histo first sample[0] %i, size %i, status %i \n", HOST_hfirstsample[0], sizeof(HOST_hfirstsample), (int)cudaCopyStatus );

  // last sample for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hlastsample), HOST_hlastsample, sizeof(HOST_hlastsample), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histo last sample FAIL, bytes %d \n", sizeof(HOST_hlastsample));
    }
  dbprintf("hipMemcpyToSymbol of histo last sample[0] %i, size %i, status %i \n", HOST_hlastsample[0], sizeof(HOST_hlastsample), (int)cudaCopyStatus );

  // rebin intervals for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hrebinintervals), HOST_hrebinintervals, sizeof(HOST_hrebinintervals), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histo rebin intervals FAIL, bytes %d \n", sizeof(HOST_hrebinintervals));
    }
  dbprintf("hipMemcpyToSymbol of histo rebin interval[0] %i, size %i, status %i \n", HOST_hrebinintervals[0], sizeof(HOST_hrebinintervals), (int)cudaCopyStatus );

  // rebin increment for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hrebinincrement), HOST_hrebinincrement, sizeof(HOST_hrebinincrement), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histo rebin increment FAIL, bytes %d \n", sizeof(HOST_hrebinincrement));
    }
  dbprintf("hipMemcpyToSymbol of histo rebin interval[0] %i, size %i, status %i \n", HOST_hrebinincrement[0], sizeof(HOST_hrebinincrement), (int)cudaCopyStatus );

  // time decimation factor for fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hdecimation), HOST_hdecimation, sizeof(HOST_hdecimation), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histo decimation FAIL, bytes %d \n", sizeof(HOST_hdecimation));
    }
  dbprintf("hipMemcpyToSymbol of histo decimation factor[0] %i, size %i, status %i \n", HOST_hdecimation[0], sizeof(HOST_hdecimation), (int)cudaCopyStatus );

  // memory offset for multi TQ-method fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hoffset), HOST_hoffset, sizeof(HOST_hoffset), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histogram memory offset FAIL, bytes %d \n", sizeof(HOST_hoffset));
    }
  dbprintf("hipMemcpyToSymbol of histogram memory offset[0,1] %i,%i,  size %i, status %i \n", HOST_hoffset[0], HOST_hoffset[1], sizeof(HOST_hoffset), (int)cudaCopyStatus );

  // time decimation factor for fill-by-fill Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_decimation), HOST_decimation, sizeof(HOST_decimation), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of decimation FAIL, bytes %d \n", sizeof(HOST_decimation));
    }
  dbprintf("hipMemcpyToSymbol of decimation factor[0,1] %i,%i,  size %i, status %i \n", HOST_decimation[0], HOST_decimation[1], sizeof(HOST_decimation), (int)cudaCopyStatus );

  // threshold values array's
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_thresholdvalues), HOST_thresholdvalues, sizeof(HOST_thresholdvalues), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of threshold values FAIL, bytes %d \n", sizeof(HOST_thresholdvalues));
    }
  dbprintf("hipMemcpyToSymbol of threshold values[0][0],[0][1] %i, %i,  size %i, status %i \n", HOST_thresholdvalues[0][0], HOST_thresholdvalues[0][1], sizeof(HOST_thresholdvalues), (int)cudaCopyStatus );
  // threshold polarities array's
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_thresholdpolarities), HOST_thresholdpolarities, sizeof(HOST_thresholdpolarities), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of bool threshold polarities FAIL, bytes %d \n", sizeof(HOST_thresholdpolarities));
    }
  dbprintf("hipMemcpyToSymbol of bool threshold polarities[0][0],[0][1] %i, %i, size %i, status %i \n", HOST_thresholdpolarities[0][0], HOST_thresholdpolarities[0][1], sizeof(HOST_thresholdpolarities), (int)cudaCopyStatus );

  // use individual threshold values
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_useindividualthresholds), HOST_useindividualthresholds, sizeof(HOST_useindividualthresholds), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of bool use threshold polarities FAIL, bytes %d \n", sizeof(HOST_useindividualthresholds));
    }
  dbprintf("hipMemcpyToSymbol of bool use threshold polarities[0] %i, size %i, status %i \n", HOST_useindividualthresholds[0], sizeof(HOST_useindividualthresholds), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_nfitislands), HOST_nfitislands, sizeof(HOST_nfitislands), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of number of fit islands FAIL, bytes %d \n", sizeof(HOST_nfitislands));
    }
  dbprintf("hipMemcpyToSymbol of number of fit islands[0,1] %i,%i,  size %i, status %i \n", HOST_nfitislands, sizeof(HOST_nfitislands), (int)cudaCopyStatus );


  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_fit_threshold), HOST_fit_threshold, sizeof(HOST_fit_threshold), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of number of fit islands FAIL, bytes %d \n", sizeof(HOST_fit_threshold));
    }
  dbprintf("hipMemcpyToSymbol of number of fit islands[0,1] %i,%i,  size %i, status %i \n", HOST_fit_threshold, sizeof(HOST_fit_threshold), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_minfittime), HOST_minfittime, sizeof(HOST_minfittime), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of number of fit islands FAIL, bytes %d \n", sizeof(HOST_minfittime));
    }
  dbprintf("hipMemcpyToSymbol of number of fit islands[0,1] %i,%i,  size %i, status %i \n", HOST_minfittime, sizeof(HOST_minfittime), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_ctag_threshold), HOST_ctag_threshold, sizeof(HOST_ctag_threshold), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG threshold FAIL, bytes %d \n", sizeof(HOST_ctag_threshold));
    }
  dbprintf("hipMemcpyToSymbol of CTAG threshold FAIL %i,%i,  size %i, status %i \n", HOST_ctag_threshold, sizeof(HOST_ctag_threshold), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_ctag_time_cut), HOST_ctag_time_cut, sizeof(HOST_ctag_time_cut), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG time cut FAIL, bytes %d \n", sizeof(HOST_ctag_time_cut));
    }
  dbprintf("hipMemcpyToSymbol of number of fit islands[0,1] %i,%i,  size %i, status %i \n", HOST_ctag_time_cut, sizeof(HOST_ctag_time_cut), (int)cudaCopyStatus );
  
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_TMask_window), HOST_TMask_window, sizeof(HOST_TMask_window),0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )  
    {
      printf("hipMemcpyToSymbol of TMask window FAIL, bytes %d \n", sizeof(HOST_TMask_window));
    }
  dbprintf("hipMemcpyToSymbol of T-method mask window %i,%i,  size %i, status %i \n", HOST_TMask_window, sizeof(HOST_TMask_window), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_mask_min), HOST_mask_min, sizeof(HOST_mask_min), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG time cut FAIL, bytes %d \n", sizeof(HOST_mask_min));
    }
  dbprintf("hipMemcpyToSymbol of minimum mask time %i,%i,  size %i, status %i \n", HOST_mask_min, sizeof(HOST_mask_min), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_mask_max), HOST_mask_max, sizeof(HOST_mask_max), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG time cut FAIL, bytes %d \n", sizeof(HOST_mask_max));
    }
  dbprintf("hipMemcpyToSymbol of maximum mask time %i,%i,  size %i, status %i \n", HOST_mask_max, sizeof(HOST_mask_max), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_mask_prescale), HOST_mask_prescale, sizeof(HOST_mask_prescale), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG time cut FAIL, bytes %d \n", sizeof(HOST_mask_prescale));
    }
  dbprintf("hipMemcpyToSymbol of minimum mask time %i,%i,  size %i, status %i \n", HOST_mask_prescale, sizeof(HOST_mask_prescale), (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_save_full_calo), HOST_save_full_calo, sizeof(HOST_save_full_calo), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of save full calo bool FAIL, bytes %d \n", sizeof(HOST_save_full_calo));
    }
  dbprintf("hipMemcpyToSymbol of save_full_calo %i,%i,  size %i, status %i \n", HOST_save_full_calo, sizeof(HOST_save_full_calo), (int)cudaCopyStatus );

cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_fill_type), HOST_fill_type, sizeof(HOST_fill_type), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of CTAG time cut FAIL, bytes %d \n", sizeof(HOST_fill_type));
    }
  dbprintf("hipMemcpyToSymbol of minimum mask time %i,%i,  size %i, status %i \n", HOST_fill_type, sizeof(HOST_fill_type), (int)cudaCopyStatus );

  // total memory size for multi TQ-method fill-summed Q-method histogram
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_hsize), HOST_hsize, sizeof(HOST_hsize), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of histogram memory size FAIL, bytes %d \n", sizeof(HOST_hsize));
    }
  dbprintf("hipMemcpyToSymbol of hsize %i,%i,  size %i, status %i \n", HOST_hsize, sizeof(HOST_hsize), (int)cudaCopyStatus );                  

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_fill_seq), HOST_fill_seq, sizeof(HOST_fill_seq), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of fill sequence FAIL, bytes %d \n", sizeof(HOST_fill_seq));
    }
  dbprintf("hipMemcpyToSymbol of fill_seq %i,%i,  size %i, status %i \n", HOST_fill_seq, sizeof(HOST_fill_seq), (int)cudaCopyStatus );                  

  // read in templates
  phaseMap pmap;
  pulseTemplate templ;
  read_template("lasertempl.bin", templ, pmap);
  std::vector<phaseMap> phaseMapVec(N_DETECTORS_MAX, pmap);
  std::vector<pulseTemplate> pTempVec(N_DETECTORS_MAX, templ);

  // copy templates to gpu
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_phase_maps), (void*)phaseMapVec.data(), phase_maps_size);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of phase maps FAIL, bytes %d \n", phase_maps_size);
    }
  dbprintf("hipMemcpyToSymbol of phase maps size %i, status %i \n", phase_maps_size, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_templates), (void*)pTempVec.data(), templates_size);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of pulse templates FAIL, bytes %d \n", templates_size);
    }
  dbprintf("hipMemcpyToSymbol of templates size %i, status %i \n", templates_size, (int)cudaCopyStatus );


  return;
}

/*
 * cuda_g2_run_kernel() calls the kernel functions that process data for each fill and copy the processed
 * data from device memory to host memory
 *
 * gpu_idata input data of continuous samples from gpu_thread to GPU
 * cpu_odata output data of T-method, Q-method to gpu_thread from GPU
 *
 */
void cuda_g2_run_kernel( unsigned char *gpu_idata, unsigned char *gpu_odata, int16_t *cpu_odata, int itq , int GPUbufferindex) {

  // hipError_t cudaCopyStatus;

  hipError_t dev_set = hipSetDevice( amc13_settings_odb.gpu_dev_id );
  if ( dev_set != hipSuccess )
    {
      printf("ERROR: (gpu_bor) acquiring CUDA device\n");
    }

  dbprintf("cuda_g2_run_kernel: TQ method index %i\n", itq);	

  // number if threads per block is optimized for GPU hardware 
  // e.g. k20 / k40 = 1024, c1060 = 512
  const int n_threads_per_block = 1024;

  // get GPU waveform length from odb parameters
  // is number of samples used for T/Q medod processing
  HOST_N_samples[itq] = tq_parameters_odb[itq].gpu_waveform_length;

  // define various block-thread grids for GPU processing functions with total
  // threads of N_samples, N_detectors, N_samples*N_detectors, etc

  // N_samples block-thread grid
  int n_blocks_nsamples = ( HOST_N_samples[itq] / n_threads_per_block ) + 1;
  if ( n_blocks_nsamples < 1 ) n_blocks_nsamples = 1;
  dim3  grid_nsamples( n_blocks_nsamples, 1, 1);
  dim3  threads_nsamples( n_threads_per_block, 1, 1);

  dbprintf("N_samples block-thread grid ::: %d samples, %d samples*detectors, %d blocks, %d threads/block, %d threads\n", 
	 HOST_N_samples[itq], HOST_N_samples[itq]*HOST_N_detectors[itq], n_blocks_nsamples, n_threads_per_block, n_blocks_nsamples*n_threads_per_block);

  // decimated N_samples block-thread grid
  int n_blocks_decimatednsamples = HOST_N_samples[itq] / n_threads_per_block / HOST_decimation[itq] + 1;
  if ( n_blocks_decimatednsamples < 1 ) n_blocks_decimatednsamples = 1;
  dim3  grid_decimatednsamples( n_blocks_decimatednsamples, 1, 1);
  dim3  threads_decimatednsamples( n_threads_per_block, 1, 1);  

  dbprintf("Decimated N_samples block-thread grid ::: %d samples, %d samples*detectors, %d blocks, %d threads/block, %d threads\n", 
	   HOST_N_samples[itq], HOST_N_samples[itq]*HOST_N_detectors[itq], n_blocks_decimatednsamples, n_threads_per_block, n_blocks_decimatednsamples*n_threads_per_block);

  // N_samples*N_detectors block-thread grid
  int n_blocks_nsamplesndetectors = HOST_N_samples[itq] * HOST_N_detectors[itq] / n_threads_per_block + 1;
  if ( n_blocks_nsamplesndetectors < 1 ) n_blocks_nsamplesndetectors = 1;
  dim3  grid_nsamplesndetectors( n_blocks_nsamplesndetectors, 1, 1);
  dim3  threads_nsamplesndetectors( n_threads_per_block, 1, 1);  

  dbprintf("N_samples*N_Detectors block-thread grid::: %d samples, %d samples*detectors, %d blocks, %d threads/block, %d threads\n", 
	   HOST_N_samples[itq], HOST_N_samples[itq]*HOST_N_detectors[itq], n_blocks_nsamplesndetectors, n_threads_per_block, n_blocks_nsamplesndetectors*n_threads_per_block);

  // N_samples*N_detectors*sizeof(int64_t)/sizeof(int16_t) block-thread grid for big-to-little endian byte re-ordering kernel function
  int n_blocks_bytereorder = ( HOST_N_samples[itq] * HOST_N_detectors[itq] / n_threads_per_block ) / ( sizeof(int64_t) / sizeof(int16_t) ) + 1;
  if ( n_blocks_bytereorder < 1 ) n_blocks_bytereorder = 1;
  dim3  grid_bytereorder( n_blocks_bytereorder, 1, 1);
  //dim3  threads_bytereorder( n_threads_per_block, 1, 1);

  dbprintf("Byte reorder block-thread grid ::: %d samples, %d samples*detectors, %d blocks, %d threads/block, %d threads\n", 
	   HOST_N_samples[itq], HOST_N_samples[itq]*HOST_N_detectors[itq], n_blocks_bytereorder, n_threads_per_block, n_blocks_bytereorder*n_threads_per_block);

  // N_detectors block-thread grid
  int n_blocks_ndetectors = 1; 
  if ( n_blocks_ndetectors < 1 ) n_blocks_ndetectors = 1;
  dim3  grid_ndetectors( n_blocks_ndetectors, 1, 1);
  dim3  threads_ndetectors(  HOST_N_detectors[itq], 1, 1);

  dbprintf("N_detectors block-thread grid ::: %d samples, %d samples*detectors, %d blocks, %d threads/block, %d threads\n", 
	   HOST_N_samples[itq], HOST_N_samples[itq]*HOST_N_detectors[itq], n_blocks_ndetectors, n_threads_per_block, n_blocks_ndetectors*n_threads_per_block);

  // GPU memory alllocation parameters
  dbprintf(" ::: start-of-kernel, size of  GPU_OBUF_SIZE 0x%08x, GPU_HIS_DATA 0x%08x, GPU_AUX_DATA 0x%08x, GPU_OUT_DATA 0x%08x \n", 
  	   GPU_OBUF_SIZE, sizeof(GPU_HIS_DATA), sizeof(GPU_AUX_DATA), sizeof(GPU_OUT_DATA) );

  // protect against cuda malloc too small
  if ( GPU_OBUF_SIZE < sizeof(GPU_HIS_DATA)+sizeof(GPU_AUX_DATA)+sizeof(GPU_OUT_DATA) ) {
    printf("GPU_OBUF_SIZE too small!!!");
    cm_msg(MERROR, __FILE__, "GPU_OBUF_SIZE too small!!!");
    return;
  }

  // measure time
  //#define TIME_MEASURE_DEF
#ifdef TIME_MEASURE_DEF
  hipEvent_t start, stop;
  hipEvent_t start_all, stop_all;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventCreate(&start_all);
  hipEventCreate(&stop_all);
  hipEventRecord(start_all, 0);
#endif // TIME_MEASURE_DEF

  // reset the output memory
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF

  // avoid zeroing of fill-by-fill histogram
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  hipMemset( auxdata, 0, ( GPU_OBUF_SIZE - sizeof(GPU_HIS_DATA) ) );

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: GPU_OBUF reset time %f ms (%i MB)\n",elapsedTime, GPU_OBUF_SIZE/1024/1024);
#endif // TIME_MEASURE_DEF

    // re-order bytes of 16-bit ADC words for big-endian 64-bit AMC13 words
    // this function is redundant since June 2016 as Rider samples payloads
    // are now little-endian rather than big-endian. 
#if 0
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_wf_be64tole16<<< grid_bytereorder, threads_bytereorder>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );

#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif
    
#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_wf_be64tole16 time %f ms n_blocks %i n_threads_per_block %i\n",elapsedTime, n_blocks_bytereorder, n_threads_per_block );
#endif // TIME_MEASURE_DEF
#endif
    
#if 1
    // calculate pedestals of detector elements (functionality controlled by pedestal option ODB parameter)
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_make_pedestals<<< grid_ndetectors, threads_ndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
    
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_make_pedestals time %f ms n_blocks %i n_threads_per_block %i\n",elapsedTime, n_blocks_ndetectors, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif 
    
#if 0
    // print function of debugging device parameters such as calo map, pedestal values
    cudaPrintfInit();
    
    dim3  grid_test( 1, 1, 1);
    dim3  threads_test( HOST_N_segments_x[itq], HOST_N_segments_y[itq], 1); 
    kernel_print_map<<< grid_test, threads_test >>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
    
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

  // if TQ processing is switched on then call TQ processing functions  
  if ( tq_parameters_odb[itq].TQ_on ){ 
    
    // function to make the distribution of ADC samples - too slow, don't use
    // kernel_wf_make_ADC<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
    
#if 1
    // calculate the calorimeter sum of all detector waveforms
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_wf_sum<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq);
    
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_wf_sum time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
    

#if 1

    //fit pulses
    if(HOST_nfitislands[itq]>0){

      if ( GPUmuonfillnumber%HOST_fit_prescale_factor[itq] == 0 ) { // prescale the fitting based on muon fill number


#ifdef TIME_MEASURE_DEF
	//start event
	hipEventRecord(start, 0);
#endif
      // zero results buffer
	hipMemset(device_fitresult, 0, result_size);

#ifdef TIME_MEASURE_DEF
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(" ::: cudaMeset for device_fitresults time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
	
	// 128 seems to give best performance
	int fit_threadsperblock = 128;
	dim3 fit_dimblock(fit_threadsperblock);
	int fit_blockspertrace =
	  ceil(static_cast<double>( HOST_N_samples[itq]) / fit_threadsperblock);
	dim3 fit_dimgrid(fit_blockspertrace);
#ifdef TIME_MEASURE_DEF
	//start event
	hipEventRecord(start, 0);
#endif
	kernel_wf_fittimes<<<fit_dimgrid, fit_dimblock>>>( (ADC_TYPE*)gpu_idata, device_fitresult, itq);

#ifdef CUDA_ERROR_CHECK
	CudaCheckError();
#endif
	
#ifdef TIME_MEASURE_DEF
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(" ::: kernel_wf_fittimes time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, fit_blockspertrace, fit_threadsperblock);
#endif // TIME_MEASURE_DEF
	
#endif
	
#if 1
	//fit pulses
	
	int nblocksx = ceil(static_cast<double>(OUTPUTARRAYLEN) / PULSESPERBLOCK);
	fit_dimgrid = dim3(nblocksx, HOST_N_detectors[itq]);
	fit_dimblock = dim3(SAMPLESPERFIT * PULSESPERBLOCK, 1);
	
#ifdef TIME_MEASURE_DEF
	//start event
	hipEventRecord(start, 0);
#endif
	//float avg = 0.;
	//for(uint16_t it=0; it<HOST_N_samples[0]; ++it){avg += static_cast<float>(gpu_idata + HOST_first_sample_index[0][0]+it);}
	//printf(" trace avg (idet=0,itq=0) =  %f\n", avg/HOST_N_samples[0]);
	//printf("HOST_N_samples[0],HOST_first_sample_index[0][0]: %d, %d\n",HOST_N_samples[0], HOST_first_sample_index[0][0]);
	kernel_wf_fitenergies<<< fit_dimgrid, fit_dimblock>>>( (ADC_TYPE*)gpu_idata, device_fitresult, itq);

#ifdef CUDA_ERROR_CHECK
	CudaCheckError();
#endif
	
#ifdef TIME_MEASURE_DEF
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(" ::: kernel_wf_fitenergies time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, nblocksx*HOST_N_detectors[itq], SAMPLESPERFIT * PULSESPERBLOCK);
#endif // TIME_MEASURE_DEF

      } // prescale fitting

    } // if fitting on
#endif
    
    bool tmask_prescale = false;
    if(GPUmuonfillnumber%HOST_mask_prescale[itq])
      tmask_prescale = true;
    
    if ( tq_parameters_odb[itq].save_full_calo || tq_parameters_odb[itq].save_truncated_calo )  { // save full or truncated calo islands 

      printf(" ::: Process CALO islands\n");
      
#if 1
      // find samples that trigger (functionality controlled by threshold option ODB parameter)
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_wf_trigger<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq, tmask_prescale);
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_wf_trigger time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
#if 1
      // extend islands by a predefined number of pre-samples and post-samples
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_extend_islands<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_extend_island time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
#if 1
      // find contiguous islands 
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_find_islands<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_find_island time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
#if 1
      // make island data structure of island time, length and sample values
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_make_islands<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_make_island time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif

    } else { // save individual xtal islands
      
      printf(" ::: Process XTAL islands\n");
#if 1
      // find samples that trigger (functionality controlled by threshold option ODB parameter)
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_wf_xtaltrigger<<< grid_nsamples, threads_nsamples>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq, tmask_prescale);
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_wf_xtaltrigger time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
#if 1
      // extend islands by a predefined number of pre-samples and post-samples
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_extend_xtalislands<<< grid_nsamplesndetectors, threads_nsamplesndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_xtalextend_island time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamplesndetectors, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
      if ( HOST_save_xtal_border[itq] ) { // save full calo islands 
	
#if 1
	// border islands identfies the border samples around the trigger samples 
#ifdef TIME_MEASURE_DEF
	// start event
	hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
	
	kernel_border_xtalislands<<< grid_nsamplesndetectors, threads_nsamplesndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );

#ifdef CUDA_ERROR_CHECK
	CudaCheckError();
#endif
	
#ifdef TIME_MEASURE_DEF
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(" ::: kernel_border_xtalisland time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamplesndetectors, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
	
      } // end save xtal borders
      
#if 1
      // find contiguous islands 
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_find_xtalislands<<< grid_nsamplesndetectors, threads_nsamplesndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_find_xtalisland time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamplesndetectors, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
#if 1
      // make island data structure of island time, length and sample values
#ifdef TIME_MEASURE_DEF
      // start event
      hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
      
      kernel_make_xtalislands<<< grid_nsamplesndetectors, threads_nsamplesndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );
      
#ifdef CUDA_ERROR_CHECK
      CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
      printf(" ::: kernel_make_xtalisland time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamplesndetectors, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
      
    } // end save xtal islands
	
#if 1
    //calculate calorimeter paramter ctag 
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_calc_ctag<<< grid_nsamples, threads_nsamples>>>( gpu_odata, itq );
    
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_calc_ctag time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_nsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
    
#if 1
    // decimate the calorimeter sum waveform for Q'-method
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_decimate_sum<<< grid_decimatednsamples, threads_decimatednsamples>>>( gpu_odata, itq );
    
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_decimate_sum time %f ms n_blocks %i n_threads_per_block %i \n",elapsedTime, n_blocks_decimatednsamples, n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif
    
#if 1
    // append the pedestal data to the end of the output data (is done after islands are obtained)
#ifdef TIME_MEASURE_DEF
    // start event
    hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
    
    kernel_pedestal_store<<< grid_ndetectors, threads_ndetectors>>>( gpu_odata, itq );
    
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

#ifdef TIME_MEASURE_DEF
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf(" ::: kernel_pedestal_store time %f ms n_blocks %i n_threads_per_block %i\n",elapsedTime, n_blocks_ndetectors,  n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif 
    
  } // end of TQ_on processing
  
  if ( tq_parameters_odb[itq].store_hist ) {

#if 0
  // accumulate the fill-by-fill sum of individual waveforms for all detectors (uses a time-decimation factor from ODB)
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF  
  
  kernel_wf_fillsum<<< grid_nsamplesndetectors, threads_nsamplesndetectors>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq );

#ifdef CUDA_ERROR_CHECK
  CudaCheckError();
#endif
  
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_fillsum time %f ms n_blocks %i n_threads_per_block %i\n",elapsedTime, n_blocks_nsamplesndetectors,  n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif

#if 1
  // accumulate the fill-by-fill sum of individual waveforms for all detectors (uses a time-decimation factor from ODB)
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF  
  
 
  int isubhist = 0;
  for (isubhist =0; isubhist < HOST_hrebinintervals[itq]; isubhist++){

    // block-thread grid Qmethod sub-histograms
    int noriginalsamples = HOST_N_detectors[itq] * ( HOST_hlastsample[itq] - HOST_hfirstsample[itq] + 1 );
    int nsubsamples = noriginalsamples / HOST_hrebinintervals[itq];
    int subrebinfactor = HOST_hdecimation[itq] * pow( (double)HOST_hrebinincrement[itq], (double)isubhist );
    int nsubbins = nsubsamples / subrebinfactor;
    int n_blocks_subhists = nsubbins / n_threads_per_block + 1;
    dbprintf("::: kernel_wf_fillsum2 itq %i ndetectors %i, isubhist %i, noriginalsamples %i, nsubsamples %i, rebinfactor %i nsubbins %i n_blocks %i\n", 
	   itq, HOST_N_detectors[itq], isubhist, noriginalsamples, nsubsamples, subrebinfactor, nsubbins, n_blocks_subhists );
    dim3  grid_nsubhists( n_blocks_subhists, 1, 1);
    dim3  threads_nsubhists( n_threads_per_block, 1, 1);  

    kernel_wf_fillsum2<<< grid_nsubhists, threads_nsubhists>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata, itq, isubhist, GPUmuonfillnumber );
#ifdef CUDA_ERROR_CHECK
    CudaCheckError();
#endif

  }

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_fillsum2 time %f ms n_blocks %i n_threads_per_block %i\n",elapsedTime, n_blocks_nsamplesndetectors,  n_threads_per_block);
#endif // TIME_MEASURE_DEF
#endif

  } // end of store_hist processing

  // size of processed data from GPU
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] = 0;

#if 1
  // copy data from GPU
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF

  // copy the 32-bit size in units of 16bit words of island data from GPU. The size incorporates the 32bit island time (size 2),
  // 32bit island length (size 2), and the 16bit (samples each size 1) for all islands.
  hipMemcpy( &GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq], gpu_odata + sizeof(GPU_HIS_DATA) + sizeof(GPU_AUX_DATA), sizeof(int), 
			     hipMemcpyDeviceToHost);

  // convert the size of N islands with time stamp and island length into units of bytes and account for padding
  // if gpu_data_proc_size[itq] is odd to 32-bit boundary.
  if (GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] % 2) GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq]++;
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] *= sizeof(int16_t);

  // increment the size of GPU output data for header items not included in island data size
 
 // 32-bit parameters island data size
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] += sizeof(int);
  // 16-bit number of islands + 16-bit number of detectors
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] += sizeof(int);
  // 32-bit reserved / CTAG word
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] += sizeof(int);

  // add the size of 32-bit decimated fill-by-fill histogram
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] += HOST_N_samples[itq] / HOST_decimation[itq] * sizeof(int32_t); // old 16-bit words for fill-by-fill, decimated histo

  // add the size of array of float pedestal values
  GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] += HOST_N_detectors[itq] * sizeof(float);

  dbprintf(" ::: GPU output data size %i\n",GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq]);
  if ( GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] > gpu_data_proc_size_max )
    {
      printf("***ERROR! too large output gpu data! %i\n",GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq]);
      GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq] = 0;
    }

  hipMemcpy( cpu_odata, gpu_odata + sizeof(GPU_HIS_DATA)+ sizeof(GPU_AUX_DATA), GPU_Data_Buffer[GPUbufferindex].gpu_data_proc_size[itq], hipMemcpyDeviceToHost);

  // Device to host copy of derived data
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: copy data from GPU time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
  
  // write HOST_N_detectors to derived data after copy from GPU
  cpu_odata[3] = HOST_N_detectors[itq];

#endif

// copy fit results
  if(HOST_nfitislands[itq]>0){
#ifdef TIME_MEASURE_DEF
  hipEventRecord(start, 0);
#endif  

  hipMemcpy(host_fitresult, device_fitresult, result_size, hipMemcpyDeviceToHost);
  /*  
  for(uint segment_num = 0; segment_num< 10; ++segment_num)
  {
  	printf("host_fitresult[%u]: energy=%f, pedestal=%f \n", segment_num, host_fitresult[segment_num].fit_results[0].energy, host_fitresult[segment_num].fit_results[0].pedestal);
  }
  */

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: copy fit results from GPU time %f ms\n",elapsedTime);
#endif  
  }

#ifdef TIME_MEASURE_DEF

  hipEventRecord(stop_all, 0);
  hipEventSynchronize(stop_all);
  hipEventElapsedTime(&elapsedTime, start_all, stop_all);
  printf(" ::: CUDA kernel total elapsed time %f ms\n",elapsedTime);

  // Clean up:
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipEventDestroy(start_all);
  hipEventDestroy(stop_all);
#endif // TIME_MEASURE_DEF

  dbprintf(" ::: end-of-kernel, size of gpu_odata 0x%08x, GPU_HIS_DATA 0x%08x, GPU_AUX_DATA 0x%08x, GPU_OUT_DATA 0x%08x \n", 
  	   GPU_OBUF_SIZE, sizeof(GPU_HIS_DATA), sizeof(GPU_AUX_DATA), sizeof(GPU_OUT_DATA) );
}

#if 0
__global__
void kernel_wf_sum_make_islands( ADC_TYPE* gpu_idata, ADC* gpu_odata, itq)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int sample_nr = bid*num_threads + tid;

  CALORIMETER_DATA_BLOCK *cal_data = (CALORIMETER_DATA_BLOCK*) gpu_odata;

  // If sample is zero finish.
  // If sample is not zero, record 5 samples before and 10 after

  if ( sample_nr >= WAVEFORM_LENGTH_MAX ) return;
  
  if ( cal_data->wf_sum_thr.adc[sample_nr] == 0 ) return;
  
  if ( sample_nr > 0 && sample_nr < (WAVEFORM_LENGTH_MAX-1) )
    {
      if ( cal_data->wf_sum_thr.adc[sample_nr-1] != 0 &&  cal_data->wf_sum_thr.adc[sample_nr+1] != 0 )
	{
	  return;
	}
    }


  for (int i=1; i<7; i++)
    {    
      int s = sample_nr - i;
      if ( s>=0 )
	{
	  cal_data->wf_sum_thr.adc[s] = cal_data->wf_sum.adc[s];
	}
    }
  
  for (int i=1; i<24; i++)
    {    
      int s = sample_nr + i;
      if ( s < WAVEFORM_LENGTH_MAX )
	{
	  cal_data->wf_sum_thr.adc[s] = cal_data->wf_sum.adc[s];
	}
    }

}


__global__
void kernel_wf_sum_glue_islands(unsigned char* gpu_odata, itq )
{

  CALORIMETER_DATA_BLOCK *calo = (CALORIMETER_DATA_BLOCK*)gpu_odata;
  int16_t *adc = calo->wf_sum_thr.adc;

  unsigned int i;
  bool sample_active = false;
  unsigned int N_islands = 0;
  //unsigned int island_len = 0;  
  unsigned int sample0 = 0;
  unsigned int offset = 0;
  // @todo replace WAVEFORM_LENGTH_MAX with actual wf length
  for (i=0; i<WAVEFORM_LENGTH_MAX; i++)
    {
      int16_t val = adc[i];
      if ( val == 0 )
	{
	  if ( sample_active )
	    {
	      // finish sample
	      calo->i_info[N_islands].sample0 = sample0;
	      unsigned int island_len = i - sample0;  
	      calo->i_info[N_islands].length = island_len;
	      N_islands++;
	      sample_active = false;
	      offset += ALIGN8(ISLAND_HEADER_LEN + island_len*ADC_SAMPLE_LEN);
	    }
	}
      else
	{
	  if ( sample_active )
	    {
	      // add new sample to the island
	      //calo->
	      ;
	    }
	  else
	    {
	      // start new island
	      calo->i_info[N_islands].offset = offset;
	      sample0 = i;
	      sample_active = true;
	    }
	}
    }
  
  calo->N_islands = N_islands;
  
  calo->Islands_len_total = offset;

}



__global__
void kernel_make_islands(unsigned char* gpu_odata, , itq)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int island_nr = bid*num_threads + tid;

  CALORIMETER_DATA_BLOCK *cal_data = (CALORIMETER_DATA_BLOCK*) gpu_odata;

  unsigned int N_islands = cal_data->N_islands;

  if ( island_nr >= N_islands ) return;
  //if ( island_nr > 1 ) return;

  unsigned int sample0   = cal_data->i_info[island_nr].sample0;
  unsigned int len       = cal_data->i_info[island_nr].length;
  unsigned int offset    = cal_data->i_info[island_nr].offset;
  unsigned int len_total = cal_data->Islands_len_total;

  unsigned int iwf;
#if 0
  for (iwf=0; iwf<WAVEFORMS_NUM; iwf++)
#endif
#if 1
  for (iwf=0; iwf<1; iwf++)
#endif
    {
      /*
      unsigned char *ptr = (unsigned char*) &(cal_data->island);
      ISLAND_HEADER *island_header = (ISLAND_HEADER*)( ptr +  
						       iwf*len_total
						       + offset);
      */
      unsigned char *ptr = (unsigned char*) &(cal_data->island);
      int16_t *island = (int16_t*)( ptr + iwf*len_total + offset );
      //unsigned int *ptr_length  = ptr_sample0+1;
      //island_nr*(sizeof(ISLAND)-sizeof(unsigned short int)) + 
      //iwf*N_islands*(sizeof(ISLAND)-sizeof(unsigned short int));

      island[0] = sample0;
      island[1] = len;
      //unsigned short int *adc_tgt = (unsigned short int*)(island_header+1);
      //ptr = (unsigned char*)island_header; 
      //unsigned short int *adc_tgt = (unsigned short int*)(ptr+sizeof(ISLAND_HEADER));
      unsigned short int *adc_tgt = (unsigned short int*)(island+2);
      unsigned short int *adc_src = cal_data->wf[iwf].adc;
      //cal_data->wf_sum.adc[sample_nr] += cal_data->wf[i].adc[sample_nr];
      unsigned int i;
      for (i=0; i<len; i++)
	{
	  adc_tgt[i] = adc_src[sample0+i]; 
	  //adc_tgt[i] = i+1;//adc_src[sample0+i]; 
	}

#if 0
      island_header->length = 10;
      island_header->sample0 = 20;
#endif

    }


}

#endif







